#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) 2020, Ramona Hohl, rhohl@nvidia.com

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

/* 
    implements main simulation kernel as well as host code to launch it
*/

// includes
// ------------------
#include "propagationKernelSource.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <chrono>

#include "settings.cuh"
#include "dataStructCuda.cuh"
#include "utils.cuh"
#include "rng.cuh"
#include "propagationKernelFunctions.cuh"
#include "zOffsetHandling.cuh"
#include "wlenGeneration.cuh"
#include "scatteringAndAbsorbtionData.cuh"
// ------------------

// remark: ignored tabulate version, removed ifdef TABULATE
// also removed ifdef DOUBLEPRECISION.
// SAVE_PHOTON_HISTORY  and SAVE_ALL_PHOTONS are not define for now, i.e. commented out these snippets,
// s.t. it corresponds to the default contstructor of I3CLSimStepToPhotonConverterOpenCL

__global__ __launch_bounds__(NTHREADS_PER_BLOCK, 4) void propKernel( I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                                                                     I3CLSimPhotonCuda* __restrict__ outputPhotons, int* numHits,
                                                                     const float* wlenLut, const float* zOffsetLut, 
                                                                     const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet, 
                                                                     uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a); 

// maxNumbWOrkItems from  CL rndm arrays
void init_RDM_CUDA(int maxNumWorkitems, uint64_t* MWC_RNG_x, uint32_t* MWC_RNG_a, uint64_t** d_MWC_RNG_x,
                   uint32_t** d_MWC_RNG_a)
{
    CUDA_ERR_CHECK(hipMalloc(d_MWC_RNG_a, maxNumWorkitems * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMalloc(d_MWC_RNG_x, maxNumWorkitems * sizeof(uint64_t)));

    CUDA_ERR_CHECK(hipMemcpy(*d_MWC_RNG_a, MWC_RNG_a, maxNumWorkitems * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_ERR_CHECK(hipMemcpy(*d_MWC_RNG_x, MWC_RNG_x, maxNumWorkitems * sizeof(uint64_t), hipMemcpyHostToDevice));

    hipDeviceSynchronize();
    printf("RNG is set up on CUDA gpu %d. \n", maxNumWorkitems);
}

void launch_CudaPropogate(const I3CLSimStep* __restrict__ in_steps, int nsteps, const uint32_t maxHitIndex,
                          unsigned short* geoLayerToOMNumIndexPerStringSet, int ngeolayer,
                          I3CLSimPhotonSeries& outphotons, uint64_t* __restrict__ MWC_RNG_x,
                          uint32_t* __restrict__ MWC_RNG_a, int sizeRNG, float& totalCudaKernelTime)
{
    // set up congruental random number generator, reusing host arrays and randomService from
    // I3CLSimStepToPhotonConverterOpenCL setup.
    uint64_t* d_MWC_RNG_x;
    uint32_t* d_MWC_RNG_a;
    init_RDM_CUDA(sizeRNG, MWC_RNG_x, MWC_RNG_a, &d_MWC_RNG_x, &d_MWC_RNG_a);

    printf("nsteps total = %d but dividing into %d launches of max size %d \n", nsteps, 1, nsteps);
    unsigned short* d_geolayer;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_geolayer, ngeolayer * sizeof(unsigned short)));
    CUDA_ERR_CHECK(hipMemcpy(d_geolayer, geoLayerToOMNumIndexPerStringSet, ngeolayer * sizeof(unsigned short),
                              hipMemcpyHostToDevice));

    struct I3CLSimStepCuda* h_cudastep = (struct I3CLSimStepCuda*)malloc(nsteps * sizeof(struct I3CLSimStepCuda));

    for (int i = 0; i < nsteps; i++) {
        h_cudastep[i] = I3CLSimStep(in_steps[i]);
    }

    I3CLSimStepCuda* d_cudastep;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudastep, nsteps * sizeof(I3CLSimStepCuda)));
    CUDA_ERR_CHECK(hipMemcpy(d_cudastep, h_cudastep, nsteps * sizeof(I3CLSimStepCuda), hipMemcpyHostToDevice));

    uint32_t* d_hitIndex;
    uint32_t h_hitIndex[1];
    h_hitIndex[0] = 0;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_hitIndex, 1 * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMemcpy(d_hitIndex, h_hitIndex, 1 * sizeof(uint32_t), hipMemcpyHostToDevice));

    I3CLSimPhotonCuda* d_cudaphotons;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudaphotons, maxHitIndex * sizeof(I3CLSimPhotonCuda)));

    int numBlocks = (nsteps + NTHREADS_PER_BLOCK - 1) / NTHREADS_PER_BLOCK;
    printf("launching kernel propKernel<<< %d , %d >>>( .., nsteps=%d)  \n", numBlocks, NTHREADS_PER_BLOCK, nsteps);

    std::chrono::time_point<std::chrono::system_clock> startKernel = std::chrono::system_clock::now();
    propKernel<<<numBlocks, NTHREADS_PER_BLOCK>>>(d_hitIndex, maxHitIndex, d_geolayer, d_cudastep, nsteps,
                                                  d_cudaphotons, d_MWC_RNG_x, d_MWC_RNG_a);

    CUDA_ERR_CHECK(hipDeviceSynchronize());
    std::chrono::time_point<std::chrono::system_clock> endKernel = std::chrono::system_clock::now();
    totalCudaKernelTime = std::chrono::duration_cast<std::chrono::milliseconds>(endKernel - startKernel).count();

    CUDA_ERR_CHECK(hipMemcpy(h_hitIndex, d_hitIndex, 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));
    int numberPhotons = h_hitIndex[0];

    if (numberPhotons > maxHitIndex) {
        printf("Maximum number of photons exceeded, only receiving %" PRIu32 " of %" PRIu32 " photons", maxHitIndex,
               numberPhotons);
        numberPhotons = maxHitIndex;
    }

    // copy (max fo maxHitIndex) photons to host.
    struct I3CLSimPhotonCuda* h_cudaphotons =
        (struct I3CLSimPhotonCuda*)malloc(numberPhotons * sizeof(struct I3CLSimPhotonCuda));
    CUDA_ERR_CHECK(
        hipMemcpy(h_cudaphotons, d_cudaphotons, numberPhotons * sizeof(I3CLSimPhotonCuda), hipMemcpyDeviceToHost));

    outphotons.resize(numberPhotons);
    for (int i = 0; i < numberPhotons; i++) {
        outphotons[i] = h_cudaphotons[i].getI3CLSimPhoton();
    }

    free(h_cudastep);
    free(h_cudaphotons);
    hipFree(d_cudaphotons);
    hipFree(d_cudastep);
    hipFree(d_geolayer);
    hipFree(d_MWC_RNG_a);
    hipFree(d_MWC_RNG_x);
    printf("photon hits = %i from %i steps \n", numberPhotons, nsteps);
}

__global__ void propKernel( I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                            I3CLSimPhotonCuda* __restrict__ outputPhotons, int* numHits,
                            const float* wlenLut, const float* zOffsetLut, 
                            const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet, 
                            uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a))
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ unsigned short geoLayerToOMNumIndexPerStringSetLocal[GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE];
    __shared__ float _generateWavelength_0distYValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float _generateWavelength_0distYCumulativeValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float getWavelengthBias_dataShared[_generateWavelength_0NUM_DIST_ENTRIES];

    for (int ii = threadIdx.x; ii < GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE; ii += blockDim.x) {
        geoLayerToOMNumIndexPerStringSetLocal[ii] = geoLayerToOMNumIndexPerStringSet[ii];
    }

    for (int ii = threadIdx.x; ii < _generateWavelength_0NUM_DIST_ENTRIES; ii += blockDim.x) {
        _generateWavelength_0distYValuesShared[ii] = _generateWavelength_0distYValues[ii];
        _generateWavelength_0distYCumulativeValuesShared[ii] = _generateWavelength_0distYCumulativeValues[ii];
        getWavelengthBias_dataShared[ii] = getWavelengthBias_data[ii];
    }
    __syncthreads();
    if (i >= nsteps) return;

    // download MWC RNG state
    uint64_t real_rnd_x = MWC_RNG_x[i];
    uint32_t real_rnd_a = MWC_RNG_a[i];
    uint64_t* rnd_x = &real_rnd_x;
    uint32_t* rnd_a = &real_rnd_a;

    const I3CLSimStepCuda step = inputSteps[i];
    float4 stepDir;
    {
        const float rho = sinf(step.dirAndLengthAndBeta.x);       // sin(theta)
        stepDir = float4{rho * cosf(step.dirAndLengthAndBeta.y),  // rho*cos(phi)
                         rho * sinf(step.dirAndLengthAndBeta.y),  // rho*sin(phi)
                         cosf(step.dirAndLengthAndBeta.x),        // cos(phi)
                         ZERO};
    }

    uint32_t photonsLeftToPropagate = step.numPhotons;
    I3CLPhoton photon;
    photon.absLength = 0;
    I3CLInitialPhoton photonInitial;

    while (photonsLeftToPropagate > 0) {
        if (photon.absLength < EPSILON) {
            photonInitial = createPhoton(step, stepDir,_generateWavelength_0distYValuesShared,_generateWavelength_0distYCumulativeValuesShared, RNG_ARGS_TO_CALL);
            photon = I3CLPhoton(photonInitial);
        }

        // this block is along the lines of the PPC kernel
        float distancePropagated;
        propPhoton(photon, distancePropagated, RNG_ARGS_TO_CALL);
        bool collided = checkForCollision(photon, photonInitial, step, distancePropagated, 
                                  hitIndex, maxHitIndex, outputPhotons, geoLayerToOMNumIndexPerStringSetLocal, getWavelengthBias_dataShared);

        if (collided) {
            // get rid of the photon if we detected it
            photon.absLength = ZERO;
        }

        // absorb or scatter the photon
        if (photon.absLength < EPSILON) {
            // photon was absorbed.
            // a new one will be generated at the begin of the loop.
            --photonsLeftToPropagate;
        } else {  // photon was NOT absorbed. scatter it and re-start the loop

            updatePhotonTrack(photon, distancePropagated);
            scatterPhoton(photon, RNG_ARGS_TO_CALL);
        }
    }  // end while

    // upload MWC RNG state
    MWC_RNG_x[i] = real_rnd_x;
    MWC_RNG_a[i] = real_rnd_a;
}
