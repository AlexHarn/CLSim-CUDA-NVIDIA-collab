#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) 2020, Ramona Hohl, rhohl@nvidia.com

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// !! order matters:
#include <fstream>
#include <propagationKernelFunctions.cuh>
#include <propagationKernelSource.cuh>

hipError_t gl_err;

#define CUDA_ERR_CHECK(e)              \
    if (hipError_t(e) != hipSuccess) \
        printf("!!! Cuda Error %s in line %d \n", hipGetErrorString(hipError_t(e)), __LINE__);
#define CUDA_CHECK_CALL                     \
    gl_err = hipGetLastError();            \
    if (hipError_t(gl_err) != hipSuccess) \
        printf("!!! Cuda Error %s in line %d \n", hipGetErrorString(hipError_t(gl_err)), __LINE__ - 1);

// remark: ignored tabulate version, removed ifdef TABULATE
// also removed ifdef DOUBLEPRECISION.
// SAVE_PHOTON_HISTORY  and SAVE_ALL_PHOTONS are not define for now, i.e. commented out these snippets,
// s.t. it corresponds to the default contstructor of I3CLSimStepToPhotonConverterOpenCL

__global__ __launch_bounds__(NTHREADS_PER_BLOCK, 4) void propKernel(
    uint32_t* hitIndex,          // deviceBuffer_CurrentNumOutputPhotons
    const uint32_t maxHitIndex,  // maxNumOutputPhotons_
    const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet,
    const I3CLSimStepCuda* __restrict__ inputSteps,  // deviceBuffer_InputSteps
    int nsteps,
    I3CLSimPhotonCuda* __restrict__ outputPhotons,  // deviceBuffer_OutputPhotons

#ifdef SAVE_PHOTON_HISTORY
    float4* photonHistory,
#endif
    uint64_t* __restrict__ MWC_RNG_x, uint32_t* __restrict__ MWC_RNG_a);

// maxNumbWOrkItems from  CL rndm arrays
void init_RDM_CUDA(int maxNumWorkitems, uint64_t* MWC_RNG_x, uint32_t* MWC_RNG_a, uint64_t** d_MWC_RNG_x,
                   uint32_t** d_MWC_RNG_a)
{
    CUDA_ERR_CHECK(hipMalloc(d_MWC_RNG_a, maxNumWorkitems * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMalloc(d_MWC_RNG_x, maxNumWorkitems * sizeof(uint64_t)));

    CUDA_ERR_CHECK(hipMemcpy(*d_MWC_RNG_a, MWC_RNG_a, maxNumWorkitems * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_ERR_CHECK(hipMemcpy(*d_MWC_RNG_x, MWC_RNG_x, maxNumWorkitems * sizeof(uint64_t), hipMemcpyHostToDevice));

    hipDeviceSynchronize();
    printf("RNG is set up on CUDA gpu %d. \n", maxNumWorkitems);
}

void launch_CudaPropogate(const I3CLSimStep* __restrict__ in_steps, int nsteps, const uint32_t maxHitIndex,
                          unsigned short* geoLayerToOMNumIndexPerStringSet, int ngeolayer,
                          I3CLSimPhotonSeries& outphotons, uint64_t* __restrict__ MWC_RNG_x,
                          uint32_t* __restrict__ MWC_RNG_a, int sizeRNG, float& totalCudaKernelTime)
{
    // set up congruental random number generator, reusing host arrays and randomService from
    // I3CLSimStepToPhotonConverterOpenCL setup.
    uint64_t* d_MWC_RNG_x;
    uint32_t* d_MWC_RNG_a;
    init_RDM_CUDA(sizeRNG, MWC_RNG_x, MWC_RNG_a, &d_MWC_RNG_x, &d_MWC_RNG_a);

    printf("nsteps total = %d but dividing into %d launches of max size %d \n", nsteps, 1, nsteps);
    unsigned short* d_geolayer;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_geolayer, ngeolayer * sizeof(unsigned short)));
    CUDA_ERR_CHECK(hipMemcpy(d_geolayer, geoLayerToOMNumIndexPerStringSet, ngeolayer * sizeof(unsigned short),
                              hipMemcpyHostToDevice));

    struct I3CLSimStepCuda* h_cudastep = (struct I3CLSimStepCuda*)malloc(nsteps * sizeof(struct I3CLSimStepCuda));

    for (int i = 0; i < nsteps; i++) {
        h_cudastep[i] = I3CLSimStep(in_steps[i]);
    }

    I3CLSimStepCuda* d_cudastep;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudastep, nsteps * sizeof(I3CLSimStepCuda)));
    CUDA_ERR_CHECK(hipMemcpy(d_cudastep, h_cudastep, nsteps * sizeof(I3CLSimStepCuda), hipMemcpyHostToDevice));

    uint32_t* d_hitIndex;
    uint32_t h_hitIndex[1];
    h_hitIndex[0] = 0;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_hitIndex, 1 * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMemcpy(d_hitIndex, h_hitIndex, 1 * sizeof(uint32_t), hipMemcpyHostToDevice));

    I3CLSimPhotonCuda* d_cudaphotons;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudaphotons, maxHitIndex * sizeof(I3CLSimPhotonCuda)));

    int numBlocks = (nsteps + NTHREADS_PER_BLOCK - 1) / NTHREADS_PER_BLOCK;
    printf("launching kernel propKernel<<< %d , %d >>>( .., nsteps=%d)  \n", numBlocks, NTHREADS_PER_BLOCK, nsteps);

    std::chrono::time_point<std::chrono::system_clock> startKernel = std::chrono::system_clock::now();
    propKernel<<<numBlocks, NTHREADS_PER_BLOCK>>>(d_hitIndex, maxHitIndex, d_geolayer, d_cudastep, nsteps,
                                                  d_cudaphotons, d_MWC_RNG_x, d_MWC_RNG_a);

    CUDA_ERR_CHECK(hipDeviceSynchronize());
    std::chrono::time_point<std::chrono::system_clock> endKernel = std::chrono::system_clock::now();
    totalCudaKernelTime = std::chrono::duration_cast<std::chrono::milliseconds>(endKernel - startKernel).count();

    CUDA_ERR_CHECK(hipMemcpy(h_hitIndex, d_hitIndex, 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));
    int numberPhotons = h_hitIndex[0];

    if (numberPhotons > maxHitIndex) {
        printf("Maximum number of photons exceeded, only receiving %" PRIu32 " of %" PRIu32 " photons", maxHitIndex,
               numberPhotons);
        numberPhotons = maxHitIndex;
    }

    // copy (max fo maxHitIndex) photons to host.
    struct I3CLSimPhotonCuda* h_cudaphotons =
        (struct I3CLSimPhotonCuda*)malloc(numberPhotons * sizeof(struct I3CLSimPhotonCuda));
    CUDA_ERR_CHECK(
        hipMemcpy(h_cudaphotons, d_cudaphotons, numberPhotons * sizeof(I3CLSimPhotonCuda), hipMemcpyDeviceToHost));

    outphotons.resize(numberPhotons);
    for (int i = 0; i < numberPhotons; i++) {
        outphotons[i] = h_cudaphotons[i].getI3CLSimPhoton();
    }

    free(h_cudastep);
    free(h_cudaphotons);
    hipFree(d_cudaphotons);
    hipFree(d_cudastep);
    hipFree(d_geolayer);
    hipFree(d_MWC_RNG_a);
    hipFree(d_MWC_RNG_x);
    printf("photon hits = %i from %i steps \n", numberPhotons, nsteps);
}

/**
 * @brief Creates a single photon to be propagated
 * @param step the step to create the photon from
 * @param stepDir step direction to create the photon ( calculated in propGroup() )
 * @param _generateWavelength_0distY data needed for wavelength selection (pass pointer to global or shared data)
 * @param _generateWavelength_0distYCumulative data needed for wavelength selection (pass pointer to global or shared data) 
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL)
 */
__device__ __forceinline__ I3CLInitialPhoton createPhoton(const I3CLSimStepCuda &step, float4 stepDir, float* _generateWavelength_0distY, float* _generateWavelength_0distYCumulative, RNG_ARGS)
{
    // create a new photon
    I3CLInitialPhoton ph;
    createPhotonFromTrack(step, stepDir, RNG_ARGS_TO_CALL, ph.posAndTime, ph.dirAndWlen, _generateWavelength_0distY, _generateWavelength_0distYCumulative);
    ph.invGroupvel = 1.f / (getGroupVelocity(0, ph.dirAndWlen.w));

    // set an initial absorption length
    ph.absLength = -logf(RNG_CALL_UNIFORM_OC);
    return ph;
}

/**
 * @brief  propgates a single photon
 * @param ph the photon to propagate
 * @param distancePropagated the distance the photon was propagated during this iteration
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL)
 * @return the propagated distance
 */
__device__ __forceinline__ bool propPhoton(I3CLPhoton& ph, float& distancePropagated, RNG_ARGS)
{ 
    const float effective_z = ph.posAndTime.z - getTiltZShift(ph.posAndTime);
    const int currentPhotonLayer = min(max(findLayerForGivenZPos(effective_z), 0), MEDIUM_LAYERS - 1);
    const float photon_dz = ph.dirAndWlen.z;

    // add a correction factor to the number of absorption lengths
    // abs_lens_left before the photon is absorbed. This factor will be
    // taken out after this propagation step. Usually the factor is 1
    // and thus has no effect, but it is used in a direction-dependent
    // way for our model of ice anisotropy.
    const float abs_len_correction_factor = getDirectionalAbsLenCorrFactor(ph.dirAndWlen);
    ph.absLength *= abs_len_correction_factor;

    // the "next" medium boundary (either top or bottom, depending on
    // step direction)
    float mediumBoundary = (photon_dz < ZERO)
                                ? (mediumLayerBoundary(currentPhotonLayer))
                                : (mediumLayerBoundary(currentPhotonLayer) + (float)MEDIUM_LAYER_THICKNESS);

     // track this thing to the next scattering point
    float scaStepLeft = -logf(RNG_CALL_UNIFORM_OC);

    float currentScaLen = getScatteringLength(currentPhotonLayer, ph.dirAndWlen.w);
    float currentAbsLen = getAbsorptionLength(currentPhotonLayer, ph.dirAndWlen.w);

    float ais = (photon_dz * scaStepLeft - ((mediumBoundary - effective_z)) / currentScaLen) *
                (ONE / (float)MEDIUM_LAYER_THICKNESS);
    float aia = (photon_dz * ph.absLength - ((mediumBoundary - effective_z)) / currentAbsLen) *
                (ONE / (float)MEDIUM_LAYER_THICKNESS);

    
    // propagate through layers
    int j = currentPhotonLayer;
    if (photon_dz < 0) {
        for (; (j > 0) && (ais < ZERO) && (aia < ZERO);
                mediumBoundary -= (float)MEDIUM_LAYER_THICKNESS,
                currentScaLen = getScatteringLength(j, ph.dirAndWlen.w),
                currentAbsLen = getAbsorptionLength(j, ph.dirAndWlen.w), ais += 1.f / (currentScaLen),
                aia += 1.f / (currentAbsLen))
            --j;
    } else {
        for (; (j < MEDIUM_LAYERS - 1) && (ais > ZERO) && (aia > ZERO);
                mediumBoundary += (float)MEDIUM_LAYER_THICKNESS,
                currentScaLen = getScatteringLength(j, ph.dirAndWlen.w),
                currentAbsLen = getAbsorptionLength(j, ph.dirAndWlen.w), ais -= 1.f / (currentScaLen),
                aia -= 1.f / (currentAbsLen))
            ++j;
    }

    float distanceToAbsorption;
    if ((currentPhotonLayer == j) || ((my_fabs(photon_dz)) < EPSILON)) {
        distancePropagated = scaStepLeft * currentScaLen;
        distanceToAbsorption = ph.absLength * currentAbsLen;
    } else {
        const float recip_photon_dz = 1.f / (photon_dz);
        distancePropagated =
            (ais * ((float)MEDIUM_LAYER_THICKNESS) * currentScaLen + mediumBoundary - effective_z) *
            recip_photon_dz;
        distanceToAbsorption =
            (aia * ((float)MEDIUM_LAYER_THICKNESS) * currentAbsLen + mediumBoundary - effective_z) *
            recip_photon_dz;
    }

    // get overburden for distance i.e. check if photon is absorbed
    if (distanceToAbsorption < distancePropagated) {
        distancePropagated = distanceToAbsorption;
        ph.absLength = ZERO;
        return true;
    } else {
        ph.absLength = (distanceToAbsorption - distancePropagated) / currentAbsLen;
        
        // hoist the correction factor back out of the absorption length
        ph.absLength = ph.absLength / abs_len_correction_factor;
        return false;
    }

}

/**
 * @brief moves a photon along its track
 * @param ph the photon to move
 * @param distancePropagated the distance the photon was propagated this iteration
 */
__device__ __forceinline__  void updatePhotonTrack(I3CLPhoton& ph, float distancePropagated)
{
        ph.posAndTime.x += ph.dirAndWlen.x * distancePropagated;
        ph.posAndTime.y += ph.dirAndWlen.y * distancePropagated;
        ph.posAndTime.z += ph.dirAndWlen.z * distancePropagated;
        ph.posAndTime.w += ph.invGroupvel * distancePropagated;
        ph.totalPathLength += distancePropagated;
}

/**
 * @brief scatters a photon
 * @param ph the photon to scatter
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL) 
 */
__device__ __forceinline__  void scatterPhoton(I3CLPhoton& ph, RNG_ARGS)
{
     // optional direction transformation (for ice anisotropy)
    transformDirectionPreScatter(ph.dirAndWlen);

    // choose a scattering angle
    const float cosScatAngle = makeScatteringCosAngle(RNG_ARGS_TO_CALL);
    const float sinScatAngle = sqrt(ONE - sqr(cosScatAngle));

    // change the current direction by that angle
    scatterDirectionByAngle(cosScatAngle, sinScatAngle, ph.dirAndWlen, RNG_CALL_UNIFORM_CO);

    // optional direction transformation (for ice anisotropy)
    transformDirectionPostScatter(ph.dirAndWlen);

    ++ph.numScatters;
}

__global__ void propKernel(uint32_t* hitIndex,          // deviceBuffer_CurrentNumOutputPhotons
                           const uint32_t maxHitIndex,  // maxNumOutputPhotons_
                           const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet,
                           const I3CLSimStepCuda* __restrict__ inputSteps,  // deviceBuffer_InputSteps
                           int nsteps,
                           I3CLSimPhotonCuda* __restrict__ outputPhotons,  // deviceBuffer_OutputPhotons
                           uint64_t* __restrict__ MWC_RNG_x, uint32_t* __restrict__ MWC_RNG_a)
{
#ifndef FUNCTION_getGroupVelocity_DOES_NOT_DEPEND_ON_LAYER
#error This kernel only works with a constant group velocity (constant w.r.t. layers)
#endif

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ unsigned short geoLayerToOMNumIndexPerStringSetLocal[GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE];
    __shared__ float _generateWavelength_0distYValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float _generateWavelength_0distYCumulativeValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float getWavelengthBias_dataShared[_generateWavelength_0NUM_DIST_ENTRIES];

    for (int ii = threadIdx.x; ii < GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE; ii += blockDim.x) {
        geoLayerToOMNumIndexPerStringSetLocal[ii] = geoLayerToOMNumIndexPerStringSet[ii];
    }

    for (int ii = threadIdx.x; ii < _generateWavelength_0NUM_DIST_ENTRIES; ii += blockDim.x) {
        _generateWavelength_0distYValuesShared[ii] = _generateWavelength_0distYValues[ii];
        _generateWavelength_0distYCumulativeValuesShared[ii] = _generateWavelength_0distYCumulativeValues[ii];
        getWavelengthBias_dataShared[ii] = getWavelengthBias_data[ii];
    }
    __syncthreads();
    if (i >= nsteps) return;

    // download MWC RNG state
    uint64_t real_rnd_x = MWC_RNG_x[i];
    uint32_t real_rnd_a = MWC_RNG_a[i];
    uint64_t* rnd_x = &real_rnd_x;
    uint32_t* rnd_a = &real_rnd_a;

    const I3CLSimStepCuda step = inputSteps[i];
    float4 stepDir;
    {
        const float rho = sinf(step.dirAndLengthAndBeta.x);       // sin(theta)
        stepDir = float4{rho * cosf(step.dirAndLengthAndBeta.y),  // rho*cos(phi)
                         rho * sinf(step.dirAndLengthAndBeta.y),  // rho*sin(phi)
                         cosf(step.dirAndLengthAndBeta.x),        // cos(phi)
                         ZERO};
    }

    uint32_t photonsLeftToPropagate = step.numPhotons;
    float abs_lens_left = ZERO;
    float abs_lens_initial = ZERO;

    float4 photonStartPosAndTime;
    float4 photonStartDirAndWlen;
    float4 photonPosAndTime;
    float4 photonDirAndWlen;
    uint32_t photonNumScatters = 0;
    float photonTotalPathLength = ZERO;

    float inv_groupvel = ZERO;

    while (photonsLeftToPropagate > 0) {
        if (abs_lens_left < EPSILON) {
            // create a new photon
            createPhotonFromTrack(step, stepDir, RNG_ARGS_TO_CALL, photonPosAndTime, photonDirAndWlen,
                                  _generateWavelength_0distYValuesShared,
                                  _generateWavelength_0distYCumulativeValuesShared);

            // save the start position and time
            photonStartPosAndTime = photonPosAndTime;
            photonStartDirAndWlen = photonDirAndWlen;

            photonNumScatters = 0;
            photonTotalPathLength = ZERO;

            inv_groupvel = 1.f / (getGroupVelocity(0, photonDirAndWlen.w));

            // the photon needs a lifetime. determine distance to next scatter and
            // absorption (this is in units of absorption/scattering lengths)
            abs_lens_initial = -logf(RNG_CALL_UNIFORM_OC);
            abs_lens_left = abs_lens_initial;
        }

        // this block is along the lines of the PPC kernel
        float distancePropagated;
        {
            const float effective_z = photonPosAndTime.z - getTiltZShift(photonPosAndTime);
            int currentPhotonLayer = min(max(findLayerForGivenZPos(effective_z), 0), MEDIUM_LAYERS - 1);
            const float photon_dz = photonDirAndWlen.z;

            // add a correction factor to the number of absorption lengths
            // abs_lens_left before the photon is absorbed. This factor will be taken
            // out after this propagation step. Usually the factor is 1 and thus has
            // no effect, but it is used in a direction-dependent way for our model of
            // ice anisotropy.

            const float abs_len_correction_factor = getDirectionalAbsLenCorrFactor(photonDirAndWlen);
            abs_lens_left *= abs_len_correction_factor;

            // the "next" medium boundary (either top or bottom, depending on step
            // direction)
            float mediumBoundary = (photon_dz < ZERO)
                                       ? (mediumLayerBoundary(currentPhotonLayer))
                                       : (mediumLayerBoundary(currentPhotonLayer) + (float)MEDIUM_LAYER_THICKNESS);

            // track this thing to the next scattering point
            float sca_step_left = -logf(RNG_CALL_UNIFORM_OC);

            float currentScaLen = getScatteringLength(currentPhotonLayer, photonDirAndWlen.w);
            float currentAbsLen = getAbsorptionLength(currentPhotonLayer, photonDirAndWlen.w);

            float ais = (photon_dz * sca_step_left - ((mediumBoundary - effective_z)) / currentScaLen) *
                        (ONE / (float)MEDIUM_LAYER_THICKNESS);
            float aia = (photon_dz * abs_lens_left - ((mediumBoundary - effective_z)) / currentAbsLen) *
                        (ONE / (float)MEDIUM_LAYER_THICKNESS);

            // propagate through layers
            int j = currentPhotonLayer;
            if (photon_dz < 0) {
                for (; (j > 0) && (ais < ZERO) && (aia < ZERO);
                     mediumBoundary -= (float)MEDIUM_LAYER_THICKNESS,
                     currentScaLen = getScatteringLength(j, photonDirAndWlen.w),
                     currentAbsLen = getAbsorptionLength(j, photonDirAndWlen.w), ais += 1.f / (currentScaLen),
                     aia += 1.f / (currentAbsLen))
                    --j;
            } else {
                for (; (j < MEDIUM_LAYERS - 1) && (ais > ZERO) && (aia > ZERO);
                     mediumBoundary += (float)MEDIUM_LAYER_THICKNESS,
                     currentScaLen = getScatteringLength(j, photonDirAndWlen.w),
                     currentAbsLen = getAbsorptionLength(j, photonDirAndWlen.w), ais -= 1.f / (currentScaLen),
                     aia -= 1.f / (currentAbsLen))
                    ++j;
            }

            float distanceToAbsorption;
            if ((currentPhotonLayer == j) || ((my_fabs(photon_dz)) < EPSILON)) {
                distancePropagated = sca_step_left * currentScaLen;
                distanceToAbsorption = abs_lens_left * currentAbsLen;
            } else {
                const float recip_photon_dz = 1.f / (photon_dz);
                distancePropagated =
                    (ais * ((float)MEDIUM_LAYER_THICKNESS) * currentScaLen + mediumBoundary - effective_z) *
                    recip_photon_dz;
                distanceToAbsorption =
                    (aia * ((float)MEDIUM_LAYER_THICKNESS) * currentAbsLen + mediumBoundary - effective_z) *
                    recip_photon_dz;
            }

            // get overburden for distance
            if (distanceToAbsorption < distancePropagated) {
                distancePropagated = distanceToAbsorption;
                abs_lens_left = ZERO;
            } else {
                abs_lens_left = (distanceToAbsorption - distancePropagated) / currentAbsLen;
            }

            // hoist the correction factor back out of the absorption length
            abs_lens_left = (abs_lens_left) / abs_len_correction_factor;
        }

        // the photon is now either being absorbed or scattered.
        // Check for collisions in its way
        bool collided = checkForCollision(
            photonPosAndTime, photonDirAndWlen, getWavelengthBias_dataShared, inv_groupvel, photonTotalPathLength,
            photonNumScatters, abs_lens_initial - abs_lens_left, photonStartPosAndTime, photonStartDirAndWlen, step,
            distancePropagated, hitIndex, maxHitIndex, outputPhotons, geoLayerToOMNumIndexPerStringSetLocal);

        if (collided) {
            // get rid of the photon if we detected it
            abs_lens_left = ZERO;
        }

        // update the track to its next position
        photonPosAndTime.x += photonDirAndWlen.x * distancePropagated;
        photonPosAndTime.y += photonDirAndWlen.y * distancePropagated;
        photonPosAndTime.z += photonDirAndWlen.z * distancePropagated;
        photonPosAndTime.w += inv_groupvel * distancePropagated;
        photonTotalPathLength += distancePropagated;

        // absorb or scatter the photon
        if (abs_lens_left < EPSILON) {
            // photon was absorbed.
            // a new one will be generated at the begin of the loop.
            --photonsLeftToPropagate;
        } else {  // photon was NOT absorbed. scatter it and re-start the loop
            // optional direction transformation (for ice anisotropy)
            transformDirectionPreScatter(photonDirAndWlen);

            // choose a scattering angle
            const float cosScatAngle = makeScatteringCosAngle(RNG_ARGS_TO_CALL);
            const float sinScatAngle = sqrt(ONE - sqr(cosScatAngle));

            // change the current direction by that angle
            scatterDirectionByAngle(cosScatAngle, sinScatAngle, photonDirAndWlen, RNG_CALL_UNIFORM_CO);

            // optional direction transformation (for ice anisotropy)
            transformDirectionPostScatter(photonDirAndWlen);

            ++photonNumScatters;
        }
    }  // end while

    // upload MWC RNG state
    MWC_RNG_x[i] = real_rnd_x;
    MWC_RNG_a[i] = real_rnd_a;
}
