#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) 2020, Ramona Hohl, rhohl@nvidia.com

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

/* 
    implements main simulation kernel as well as host code to launch it
*/

// includes
// ------------------
#include "propagationKernelSource.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <chrono>

#include "settings.cuh"
#include "dataStructCuda.cuh"
#include "utils.cuh"
#include "rng.cuh"
#include "propagationKernelFunctions.cuh"
#include "zOffsetHandling.cuh"
#include "wlenGeneration.cuh"
#include "scatteringAndAbsorbtionData.cuh"
// ------------------

// remark: ignored tabulate version, removed ifdef TABULATE
// also removed ifdef DOUBLEPRECISION.
// SAVE_PHOTON_HISTORY  and SAVE_ALL_PHOTONS are not define for now, i.e. commented out these snippets,
// s.t. it corresponds to the default contstructor of I3CLSimStepToPhotonConverterOpenCL

__global__ __launch_bounds__(NTHREADS_PER_BLOCK, 4) void propKernel( I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                                                                    uint32_t* hitIndex, uint32_t maxHitIndex, I3CLSimPhotonCuda* __restrict__ outputPhotons,
                                                                    const float* wlenLut, const float* zOffsetLut, 
                                                                    const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet, 
                                                                    uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a); 

void launch_CudaPropogate(const I3CLSimStep* __restrict__ in_steps, int nsteps, const uint32_t maxHitIndex,
                          unsigned short* geoLayerToOMNumIndexPerStringSet, int ngeolayer,
                          I3CLSimPhotonSeries& outphotons, uint64_t* __restrict__ MWC_RNG_x,
                          uint32_t* __restrict__ MWC_RNG_a, int sizeRNG, float& totalCudaKernelTime)
{
    // setup the rng
    uint64_t* d_MWC_RNG_x;
    uint32_t* d_MWC_RNG_a;
    initMWCRng(sizeRNG, MWC_RNG_x, MWC_RNG_a, &d_MWC_RNG_x, &d_MWC_RNG_a);

    printf("nsteps total = %d but dividing into %d launches of max size %d \n", nsteps, 1, nsteps);

    // upload "geo layer per string set" data 
    unsigned short* d_geolayer;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_geolayer, ngeolayer * sizeof(unsigned short)));
    CUDA_ERR_CHECK(hipMemcpy(d_geolayer, geoLayerToOMNumIndexPerStringSet, ngeolayer * sizeof(unsigned short),
                              hipMemcpyHostToDevice));

    // convert and upload steps
    I3CLSimStepCuda* h_cudastep = (I3CLSimStepCuda*)malloc(nsteps * sizeof(struct I3CLSimStepCuda));
    for (int i = 0; i < nsteps; i++) {
        h_cudastep[i] = I3CLSimStepCuda(in_steps[i]);
    }
    I3CLSimStepCuda* d_cudastep;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudastep, nsteps * sizeof(I3CLSimStepCuda)));
    CUDA_ERR_CHECK(hipMemcpy(d_cudastep, h_cudastep, nsteps * sizeof(I3CLSimStepCuda), hipMemcpyHostToDevice));

    // allocate storage to store hits
    uint32_t* d_hitIndex;
    uint32_t h_hitIndex[1];
    h_hitIndex[0] = 0;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_hitIndex, 1 * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMemcpy(d_hitIndex, h_hitIndex, 1 * sizeof(uint32_t), hipMemcpyHostToDevice));

    I3CLSimPhotonCuda* d_cudaphotons;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudaphotons, maxHitIndex * sizeof(I3CLSimPhotonCuda)));

    // wlen lut
    auto wlenLut = generateWavelengthLut();
    float* d_wlenLut;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_wlenLut, WLEN_LUT_SIZE * sizeof(float)));
    CUDA_ERR_CHECK(hipMemcpy(d_wlenLut, wlenLut.data(), WLEN_LUT_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // zOffset lut
    auto zOffsetLut = generateZOffsetLut();
    float* d_zOffsetLut;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_zOffsetLut, zOffsetLut.size() * sizeof(float)));
    CUDA_ERR_CHECK(hipMemcpy(d_zOffsetLut, zOffsetLut.data(), zOffsetLut.size() * sizeof(float), hipMemcpyHostToDevice));

    // compute block number and launch
    int numBlocks = (nsteps + NTHREADS_PER_BLOCK - 1) / NTHREADS_PER_BLOCK;
    printf("launching kernel propKernel<<< %d , %d >>>( .., nsteps=%d)  \n", numBlocks, NTHREADS_PER_BLOCK, nsteps);

    std::chrono::time_point<std::chrono::system_clock> startKernel = std::chrono::system_clock::now();
    propKernel<<<numBlocks, NTHREADS_PER_BLOCK>>>(d_cudastep, nsteps, 
                                                  d_hitIndex, maxHitIndex, d_cudaphotons,
                                                  d_wlenLut, d_zOffsetLut,
                                                  d_geolayer,
                                                  d_MWC_RNG_x, d_MWC_RNG_a);

    CUDA_ERR_CHECK(hipDeviceSynchronize());
    std::chrono::time_point<std::chrono::system_clock> endKernel = std::chrono::system_clock::now();
    totalCudaKernelTime = std::chrono::duration_cast<std::chrono::milliseconds>(endKernel - startKernel).count();

    CUDA_ERR_CHECK(hipMemcpy(h_hitIndex, d_hitIndex, 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));
    int numberPhotons = h_hitIndex[0];

    if (numberPhotons > maxHitIndex) {
        printf("Maximum number of photons exceeded, only receiving %u of %u photons", maxHitIndex, numberPhotons);
        numberPhotons = maxHitIndex;
    }

    // copy (max fo maxHitIndex) photons to host.
    struct I3CLSimPhotonCuda* h_cudaphotons =
        (struct I3CLSimPhotonCuda*)malloc(numberPhotons * sizeof(struct I3CLSimPhotonCuda));
    CUDA_ERR_CHECK(
        hipMemcpy(h_cudaphotons, d_cudaphotons, numberPhotons * sizeof(I3CLSimPhotonCuda), hipMemcpyDeviceToHost));

    outphotons.resize(numberPhotons);
    for (int i = 0; i < numberPhotons; i++) {
        outphotons[i] = h_cudaphotons[i].getI3CLSimPhoton();
    }

    free(h_cudastep);
    free(h_cudaphotons);
    hipFree(d_cudaphotons);
    hipFree(d_cudastep);
    hipFree(d_geolayer);
    hipFree(d_MWC_RNG_a);
    hipFree(d_MWC_RNG_x);
    printf("photon hits = %i from %i steps \n", numberPhotons, nsteps);
}

__global__ void propKernel( I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                            uint32_t* hitIndex, uint32_t maxHitIndex, I3CLSimPhotonCuda* __restrict__ outputPhotons,
                            const float* wlenLut, const float* zOffsetLut, 
                            const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet, 
                            uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a)
{
    // copy some LUTs to shared memory for faster access
    __shared__ unsigned short geoLayerToOMNumIndexPerStringSetLocal[GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE];
    __shared__ float getWavelengthBias_dataShared[43];
    __shared__ float sharedScatteringLength[171];
    __shared__ float sharedAbsorptionADust[171];
    __shared__ float sharedAbsorptionDeltaTau[171];

    for (int i = threadIdx.x; i < 171; i += blockDim.x) {
        sharedScatteringLength[i] = scatteringLength_b400_LUT[i];
        sharedAbsorptionADust[i] = absorptionLength_aDust400_LUT[i];
        sharedAbsorptionDeltaTau[i] = absorptionLength_deltaTau_LUT[i];
    }

    for (int i = threadIdx.x; i < GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE; i += blockDim.x) {
        geoLayerToOMNumIndexPerStringSetLocal[i] = geoLayerToOMNumIndexPerStringSet[i];
    }

    for (int i = threadIdx.x; i < 43; i += blockDim.x) {
        getWavelengthBias_dataShared[i] = getWavelengthBias_data[i];
    }
    __syncthreads();

    // get thread id
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id > numSteps)
        return;

    // initialize rng
    RngType rng(rng_x[id],rng_a[id]);

    // load step and calculate direction
    const I3CLSimStepCuda step = steps[id];
    const float3 stepDir = calculateStepDir(step);

    // variables to store data about current photon
    uint32_t photonsLeftToPropagate = step.numPhotons;
    I3CLPhoton photon;
    I3CLInitialPhoton photonInitial;
    photon.absLength = 0.0f;

    // loop until all photons are done
    while (photonsLeftToPropagate > 0) {

        // if current photon is done, create a new one
        if (photon.absLength < EPSILON) {
            photonInitial = createPhoton(step, stepDir, wlenLut, rng);
            photon = I3CLPhoton(photonInitial);
        }

        // propagate through layers until scattered or absorbed
        float distanceTraveled;
        bool absorbed = propPhoton(photon, distanceTraveled, rng, sharedScatteringLength, sharedAbsorptionADust, sharedAbsorptionDeltaTau, zOffsetLut);
        
        // check for collision with DOMs, if collision has happened, the hit will be stored in outputPhotons
        bool collided = checkForCollisionOld(photon, step, distanceTraveled, 
                                  hitIndex, maxHitIndex, outputPhotons, geoLayerToOMNumIndexPerStringSetLocal, getWavelengthBias_dataShared);

        // remove photon if it is collided or absorbed
        // we get the next photon at the beginning of the loop
        if (collided || absorbed) {
            photon.absLength = 0.0f;
            --photonsLeftToPropagate;
        }
        else
        {
            // move the photon along its current direction for the distance it was propagated through the ice
            // then scatter to find a new direction vector
            updatePhotonTrack(photon, distanceTraveled);
            scatterPhoton(photon, rng);
        }
    }

    // store rng state
    rng_x[id] = rng.getState();
}
