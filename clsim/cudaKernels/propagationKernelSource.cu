#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) 2020, Hendrik Schwanekamp hschwanekamp@nvidia.com, Ramona Hohl rhohl@nvidia.com

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGSEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

/* 
    implements main simulation kernel as well as host code to launch it
*/

// includes
// ------------------
#include "propagationKernelSource.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <chrono>

#include "settings.cuh"
#include "dataStructCuda.cuh"
#include "utils.cuh"
#include "rng.cuh"
#include "propagationKernelFunctions.cuh"
#include "zOffsetHandling.cuh"
#include "wlenGeneration.cuh"
#include "scatteringAndAbsorbtionData.cuh"
// ------------------

// remark: ignored tabulate version, removed ifdef TABULATE
// also removed ifdef DOUBLEPRECISION.
// SAVE_PHOTON_HISTORY  and SAVE_ALL_PHOTONS are not define for now, i.e. commented out these snippets,
// s.t. it corresponds to the default contstructor of I3CLSimStepToPhotonConverterOpenCL

__global__ __launch_bounds__(NTHREADS_PER_BLOCK, NBLOCKS_PER_SM) void propKernel( I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                                                                    uint32_t* hitIndex, uint32_t maxHitIndex, I3CLSimPhotonCuda* __restrict__ outputPhotons,
                                                                    const float DOMOversizeFactor,
                                                                    const float* wlenLut, const float* wlenBias, const float* zOffsetLut,
                                                                    const float* scatteringLength_b400_LUT, const float* absorptionLength_aDust400_LUT,
                                                                    const float* absorptionLength_deltaTau_LUT,
                                                                    uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a); 

__global__ __launch_bounds__(NTHREADS_PER_BLOCK, NBLOCKS_PER_SM) void propKernelJobqueue(I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                                                                    uint32_t* hitIndex, uint32_t maxHitIndex, I3CLSimPhotonCuda* __restrict__ outputPhotons,
                                                                    const float* wlenLut, const float* zOffsetLut,
                                                                    uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a, int numPrimes); 

template <typename T, typename P>
void vectorToDevice(T** ptr, const std::vector<P> &data)
{
    std::vector<T> vec(data.begin(), data.end());
    CUDA_ERR_THROW(hipMalloc((void**)ptr, vec.size()*sizeof(T)));
    CUDA_ERR_THROW(hipMemcpy(*ptr, vec.data(), vec.size() * sizeof(T), hipMemcpyHostToDevice));
}

struct KernelBuffers {
    float  DOMOversizeFactor;
    float* wlenLut;
    float* wlenBias;
    float* zOffsetLut;
    float* scatteringLength_b400_LUT;
    float* absorptionLength_aDust400_LUT;
    float* absorptionLength_deltaTau_LUT;
    uint64_t *MWC_RNG_x;
    uint32_t *MWC_RNG_a;
    I3CLSimStepCuda* inputSteps;
    uint32_t numInputSteps;
    I3CLSimPhotonCuda* outputPhotons;
    uint32_t *numOutputPhotons;
    uint32_t maxHitIndex;
    hipStream_t stream;
    KernelBuffers(
        size_t maxNumWorkItems, size_t maxNumOutputPhotons,
        const float DOMOversizeFactor,
        const std::vector<double> &wavelengths,
        const std::vector<double> &wavelengthBias,
        const std::vector<double> &wavelengthPMF,
        const std::vector<double> &wavelengthCDF,
        const std::vector<double> &scatteringLength_b400,
        const std::vector<double> &absorptionLength_aDust400,
        const std::vector<double> &absorptionLength_deltaTau,
        const std::vector<uint64_t> &x, const std::vector<uint32_t> &a
    ) :
        DOMOversizeFactor(DOMOversizeFactor),
        wlenLut(nullptr),
        wlenBias(nullptr),
        zOffsetLut(nullptr),
        scatteringLength_b400_LUT(nullptr),
        absorptionLength_aDust400_LUT(nullptr),
        absorptionLength_deltaTau_LUT(nullptr),
        MWC_RNG_x(nullptr),
        MWC_RNG_a(nullptr),
        inputSteps(nullptr),
        numInputSteps(0),
        outputPhotons(nullptr),
        numOutputPhotons(nullptr),
        maxHitIndex(maxNumOutputPhotons)
    {
        {
            if (wavelengths.size() != 43) {
                throw std::runtime_error("Wavelength table must have exactly 43 entries");
            }
            if (wavelengths[0] != 2.6e-7) {
                throw std::runtime_error("Wavelength table must start at 260 nm");
            }
            if (fabs(wavelengths[1]-wavelengths[0]-1e-8) > 1e-12) {
                throw std::runtime_error("Wavelength step must be 10 nm");
            }

            auto wlen = generateWavelengthLut(wavelengthPMF.data(), wavelengthCDF.data());
            CUDA_ERR_THROW(hipMalloc((void**)&wlenLut, wlen.size()*sizeof(float)));
            CUDA_ERR_THROW(hipMemcpy(wlenLut, wlen.data(), wlen.size() * sizeof(float), hipMemcpyHostToDevice));

            std::vector<float> bias(wavelengthBias.begin(), wavelengthBias.end());
            CUDA_ERR_THROW(hipMalloc((void**)&wlenBias, bias.size()*sizeof(float)));
            CUDA_ERR_THROW(hipMemcpy(wlenBias, bias.data(), bias.size() * sizeof(float), hipMemcpyHostToDevice));
        }
        {
            if (scatteringLength_b400.size() != 171) {
                throw std::runtime_error("b400 must have 171 entries");
            }
            if (absorptionLength_aDust400.size() != 171) {
                throw std::runtime_error("b400 must have 171 entries");
            }
            if (absorptionLength_deltaTau.size() != 171) {
                throw std::runtime_error("b400 must have 171 entries");
            }
            vectorToDevice(&scatteringLength_b400_LUT, scatteringLength_b400);
            vectorToDevice(&absorptionLength_aDust400_LUT, absorptionLength_aDust400);
            vectorToDevice(&absorptionLength_deltaTau_LUT, absorptionLength_deltaTau);
        }
        {
            auto zOffset = generateZOffsetLut();
            CUDA_ERR_THROW(hipMalloc((void**)&zOffsetLut, zOffset.size() * sizeof(float)));
            CUDA_ERR_THROW(hipMemcpy(zOffsetLut, zOffset.data(), zOffset.size() * sizeof(float), hipMemcpyHostToDevice));
        }
        {
            // FIXME: add jobqueue support back
            initMWCRng(x.size(), x.data(), a.data(), &MWC_RNG_x, &MWC_RNG_a);
        }
        CUDA_ERR_THROW(hipMalloc((void**)&inputSteps, maxNumWorkItems * sizeof(I3CLSimStepCuda)));
        CUDA_ERR_THROW(hipMalloc((void**)&outputPhotons, maxNumOutputPhotons * sizeof(I3CLSimPhotonCuda)));
        CUDA_ERR_THROW(hipMalloc((void**)&numOutputPhotons, sizeof(uint32_t)));
        CUDA_ERR_THROW(hipStreamCreate(&stream));
    }

    ~KernelBuffers() {
        if (wlenLut != nullptr)
            hipFree(wlenLut);
        if (wlenBias != nullptr)
            hipFree(wlenBias);
        if (zOffsetLut != nullptr)
            hipFree(zOffsetLut);
        if (scatteringLength_b400_LUT != nullptr)
            hipFree(scatteringLength_b400_LUT);
        if (absorptionLength_aDust400_LUT != nullptr)
            hipFree(absorptionLength_aDust400_LUT);
        if (absorptionLength_deltaTau_LUT != nullptr)
            hipFree(absorptionLength_deltaTau_LUT);
        if (MWC_RNG_x != nullptr)
            hipFree(MWC_RNG_x);
        if (MWC_RNG_a != nullptr)
            hipFree(MWC_RNG_a);
        if (inputSteps != nullptr)
            hipFree(inputSteps);
        if (outputPhotons != nullptr)
            hipFree(outputPhotons);
        if (numOutputPhotons != nullptr)
            hipFree(numOutputPhotons);
        hipStreamDestroy(stream);
    }

// hide from device compilation trajectory (I3CLSimStep contains unsupported vector types)
#ifndef __CUDA_ARCH__
    void uploadSteps(const std::vector<I3CLSimStep> &steps) {
        // upload steps
        std::vector<I3CLSimStepCuda> cudaSteps(steps.size());
        for (int i = 0; i < steps.size(); i++) {
            cudaSteps[i] = I3CLSimStepCuda(steps[i]);
        }
        CUDA_ERR_THROW(hipMemcpyAsync(inputSteps, cudaSteps.data(), cudaSteps.size() * sizeof(I3CLSimStepCuda), hipMemcpyHostToDevice, stream));

        // reset end of output buffer
        uint32_t zero = 0;
        CUDA_ERR_THROW(hipMemcpyAsync(numOutputPhotons, &zero, sizeof(uint32_t), hipMemcpyHostToDevice, stream));

        CUDA_ERR_THROW(hipStreamSynchronize(stream));
        numInputSteps = cudaSteps.size();
    }
    std::vector<I3CLSimPhoton> downloadPhotons() {
        uint32_t numberPhotons;
        CUDA_ERR_THROW(hipMemcpyAsync(&numberPhotons, numOutputPhotons, 1 * sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
        CUDA_ERR_THROW(hipStreamSynchronize(stream));
        std::vector<I3CLSimPhotonCuda> cudaPhotons(numberPhotons);
        std::vector<I3CLSimPhoton> photons(numberPhotons);
        CUDA_ERR_THROW(hipMemcpyAsync(cudaPhotons.data(), outputPhotons, numberPhotons * sizeof(I3CLSimPhotonCuda), hipMemcpyDeviceToHost, stream));
        CUDA_ERR_THROW(hipStreamSynchronize(stream));
        for (int i = 0; i < numberPhotons; i++) {
            photons[i] = cudaPhotons[i].getI3CLSimPhoton();
        }
        return photons;
    }
#endif
};

Kernel::Kernel(
    int device,
    size_t maxNumWorkItems,
    size_t maxNumOutputPhotons,
    const float DOMOversizeFactor,
    const std::vector<double> &wavelengths,
    const std::vector<double> &wavelengthBias,
    const std::vector<double> &wavelengthPMF,
    const std::vector<double> &wavelengthCDF,
    const std::vector<double> &scatteringLength_b400,
    const std::vector<double> &absorptionLength_aDust400,
    const std::vector<double> &absorptionLength_deltaTau,
    const std::vector<uint64_t> &x,
    const std::vector<uint32_t> &a
) : impl(
        new KernelBuffers(
            maxNumWorkItems,
            maxNumOutputPhotons,
            DOMOversizeFactor,
            wavelengths,
            wavelengthBias,
            wavelengthPMF,
            wavelengthCDF,
            scatteringLength_b400,
            absorptionLength_aDust400,
            absorptionLength_deltaTau,
            x,
            a
        )
    )
{
    CUDA_ERR_THROW(hipSetDevice(device));
}

// dtor here, where KernelBuffers is complete
Kernel::~Kernel() {}

#ifndef __CUDA_ARCH__
void Kernel::uploadSteps(const std::vector<I3CLSimStep> &steps) { impl->uploadSteps(steps); }
std::vector<I3CLSimPhoton> Kernel::downloadPhotons() { return impl->downloadPhotons(); }
size_t Kernel::execute() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, impl->stream);
#ifdef USE_JOBQUEUE
    propKernelJobqueue<<<numBlocks, NTHREADS_PER_BLOCK, 0, impl->stream >>>(impl->inputSteps, impl->numInputSteps,
                                                impl->numOutputPhotons, impl->maxHitIndex, impl->outputPhotons,
                                                impl->wlenLut, impl->zOffsetLut,
                                                impl->MWC_RNG_x, impl->MWC_RNG_a, sizeRNG);
#else
    int numBlocks = (impl->numInputSteps + NTHREADS_PER_BLOCK - 1) / NTHREADS_PER_BLOCK;
    propKernel<<<numBlocks, NTHREADS_PER_BLOCK, 0, impl->stream >>>(impl->inputSteps, impl->numInputSteps,
                                                impl->numOutputPhotons, impl->maxHitIndex, impl->outputPhotons,
                                                impl->DOMOversizeFactor,
                                                impl->wlenLut, impl->wlenBias, impl->zOffsetLut,
                                                impl->scatteringLength_b400_LUT, impl->absorptionLength_aDust400_LUT,
                                                impl->absorptionLength_deltaTau_LUT,
                                                impl->MWC_RNG_x, impl->MWC_RNG_a);
#endif
    CUDA_ERR_THROW(hipGetLastError());
    hipEventRecord(stop, impl->stream);
    CUDA_ERR_THROW(hipEventSynchronize(stop));

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return size_t(floor(milliseconds*1e6f));
}
#endif

__global__ void propKernel( I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                            uint32_t* hitIndex, uint32_t maxHitIndex, I3CLSimPhotonCuda* __restrict__ outputPhotons,
                            const float DOMOversizeFactor,
                            const float* wlenLut, const float* getWavelengthBias_data, const float* zOffsetLut,
                            const float* scatteringLength_b400_LUT, const float* absorptionLength_aDust400_LUT,
                            const float* absorptionLength_deltaTau_LUT,
                            uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a)
{
    #ifdef SHARED_WLEN
        __shared__ float sharedWlenLut[WLEN_LUT_SIZE];
        for (int i = threadIdx.x; i < WLEN_LUT_SIZE; i += blockDim.x) {
            sharedWlenLut[i] = wlenLut[i];
        }
        const float* wlenLutPointer = sharedWlenLut;
    #else
        const float* wlenLutPointer = wlenLut;
    #endif

    #ifdef SHARED_ICE_PROPERTIES
        __shared__ float sharedScatteringLength[171];
        __shared__ float sharedAbsorptionADust[171];
        __shared__ float sharedAbsorptionDeltaTau[171];
        for (int i = threadIdx.x; i < 171; i += blockDim.x) {
            sharedScatteringLength[i] = scatteringLength_b400_LUT[i];
            sharedAbsorptionADust[i] = absorptionLength_aDust400_LUT[i];
            sharedAbsorptionDeltaTau[i] = absorptionLength_deltaTau_LUT[i];
        }
        const float* scatteringLutPointer = sharedScatteringLength;
        const float* absorbtionLutPointer = sharedAbsorptionADust;
        const float* absorbtionDeltaTauLutPointer = sharedAbsorptionDeltaTau;
    #else
        const float* scatteringLutPointer = scatteringLength_b400_LUT;
        const float* absorbtionLutPointer = absorptionLength_aDust400_LUT;
        const float* absorbtionDeltaTauLutPointer = absorptionLength_deltaTau_LUT;
    #endif

    #ifdef SHARED_WLEN_BIAS
        __shared__ float getWavelengthBias_dataShared[43];
        for (int i = threadIdx.x; i < 43; i += blockDim.x) {
            getWavelengthBias_dataShared[i] = getWavelengthBias_data[i];
        }
        const float* wlenBiasLutPointer = getWavelengthBias_dataShared;
    #else
        const float* wlenBiasLutPointer = getWavelengthBias_data;
    #endif

    #ifdef SHARED_NUM_INDEX_STRING_SET
        __shared__ unsigned short geoLayerToOMNumIndexPerStringSetLocal[GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE];
        for (int i = threadIdx.x; i < GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE; i += blockDim.x) {
            geoLayerToOMNumIndexPerStringSetLocal[i] = geoLayerToOMNumIndexPerStringSet[i];
        }
        const unsigned short* numIndexStringSetPointer = geoLayerToOMNumIndexPerStringSetLocal;
    #else
        const unsigned short* numIndexStringSetPointer = geoLayerToOMNumIndexPerStringSet;
    #endif

    #ifdef SHARED_COLLISION_GRID_DATA
        __shared__ unsigned short geoCellIndex0shared[GEO_CELL_NUM_X_0 * GEO_CELL_NUM_Y_0];
        __shared__ unsigned short geoCellIndex1shared[GEO_CELL_NUM_X_1 * GEO_CELL_NUM_Y_1];
        for (int i = threadIdx.x; i < GEO_CELL_NUM_X_0 * GEO_CELL_NUM_Y_0; i += blockDim.x) {
            geoCellIndex0shared[i] = geoCellIndex_0[i];
        }
        for (int i = threadIdx.x; i < GEO_CELL_NUM_X_1 * GEO_CELL_NUM_Y_1; i += blockDim.x) {
            geoCellIndex1shared[i] = geoCellIndex_1[i];
        }
        const unsigned short* geoCellIndex0Pointer = geoCellIndex0shared;
        const unsigned short* geoCellIndex1Pointer = geoCellIndex1shared;
    #else
        const unsigned short* geoCellIndex0Pointer = geoCellIndex_0;
        const unsigned short* geoCellIndex1Pointer = geoCellIndex_1;
    #endif

    #ifdef SHARED_STRING_DATA
        __shared__ unsigned char geoStringInSetShared[NUM_STRINGS];
        __shared__ unsigned short geoLayerNumShared[GEO_LAYER_STRINGSET_NUM];
        __shared__ float geoLayerStartZShared[GEO_LAYER_STRINGSET_NUM];
        __shared__ float geoLayerHeightShared[GEO_LAYER_STRINGSET_NUM];
        for (int i = threadIdx.x; i < NUM_STRINGS; i += blockDim.x) {
            geoStringInSetShared[i] = geoStringInStringSet[i];
        }
        for (int i = threadIdx.x; i < GEO_LAYER_STRINGSET_NUM; i += blockDim.x) {
            geoLayerNumShared[i] = geoLayerNum[i];
            geoLayerStartZShared[i] = geoLayerStartZ[i];
            geoLayerHeightShared[i] = geoLayerHeight[i];
        }
        const unsigned char* geoStringInSetPointer = geoStringInSetShared;
        const unsigned short* geoLayerNumPointer = geoLayerNumShared;
        const float* geoLayerStartZPointer = geoLayerStartZShared;
        const float* geoLayerHeightPointer = geoLayerHeightShared;
    #else
        const unsigned char* geoStringInSetPointer = geoStringInStringSet;
        const unsigned short* geoLayerNumPointer = geoLayerNum;
        const float* geoLayerStartZPointer = geoLayerStartZ;
        const float* geoLayerHeightPointer = geoLayerHeight;
    #endif

    #ifdef SHARED_STRING_POSITIONS
        __shared__ float geoStringPosXShared[NUM_STRINGS];
        __shared__ float geoStringPosYShared[NUM_STRINGS];
        __shared__ float geoStringMinZShared[NUM_STRINGS];
        __shared__ float geoStringMaxZShared[NUM_STRINGS];
        for (int i = threadIdx.x; i < NUM_STRINGS; i += blockDim.x) {
            geoStringPosXShared[i] = geoStringPosX[i];
            geoStringPosYShared[i] = geoStringPosY[i];
            geoStringMinZShared[i] = geoStringMinZ[i];
            geoStringMaxZShared[i] = geoStringMaxZ[i];
        }
        const float* geoStringPosXPointer = geoStringPosXShared;
        const float* geoStringPosYPointer = geoStringPosYShared;
        const float* geoStringMinZPointer = geoStringMinZShared;
        const float* geoStringMaxZPointer = geoStringMaxZShared;
    #else
        const float* geoStringPosXPointer = geoStringPosX;
        const float* geoStringPosYPointer = geoStringPosY;
        const float* geoStringMinZPointer = geoStringMinZ;
        const float* geoStringMaxZPointer = geoStringMaxZ;
    #endif

    __syncthreads();

    // get thread id
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id > numSteps)
        return;

    // initialize rng
    RngType rng(rng_x[id],rng_a[id]);

    // load step and calculate direction
    const I3CLSimStepCuda step = steps[id];
    const float3 stepDir = calculateStepDir(step);

    // variables to store data about current photon
    uint32_t photonsLeftToPropagate = step.numPhotons;
    I3CLPhoton photon;
    photon.absLength = 0.0f;

    // loop until all photons are done
    while (photonsLeftToPropagate > 0) {

        // if current photon is done, create a new one
        if (photon.absLength < EPSILON) {
            photon = createPhoton(step, stepDir, wlenLutPointer, rng);
        }

        // propagate through layers until scattered or absorbed
        float distanceTraveled;
        bool absorbed = propPhoton(photon, distanceTraveled, rng, scatteringLutPointer, absorbtionLutPointer, absorbtionDeltaTauLutPointer, zOffsetLut);

        // check for collision with DOMs, if collision has happened, the hit will be stored in outputPhotons
        bool collided = checkForCollisionOld(photon, step, distanceTraveled, 
                                  hitIndex, maxHitIndex, outputPhotons, DOMOversizeFactor,
                                  numIndexStringSetPointer, wlenBiasLutPointer, 
                                  geoCellIndex0Pointer, geoCellIndex1Pointer, geoStringInSetPointer, geoLayerNumPointer, geoLayerStartZPointer, geoLayerHeightPointer,
                                  geoStringPosXPointer, geoStringPosYPointer, geoStringMinZPointer, geoStringMaxZPointer);

        // remove photon if it is collided or absorbed
        // we get the next photon at the beginning of the loop
        if (collided || absorbed) {
            photon.absLength = 0.0f;
            --photonsLeftToPropagate;
        }
        else
        {
            // move the photon along its current direction for the distance it was propagated through the ice
            // then scatter to find a new direction vector
            updatePhotonTrack(photon, distanceTraveled);
            scatterPhoton(photon, rng);
        }
    }

    // store rng state
    rng_x[id] = rng.getState();
}

// thread_block_tile::meta_group_rank() requires CUDA 11
//#if CUDA_VERSION >= 11000
#if CUDA_VERSION >= 99999999999  // TODO: add jobqueue support back
/**
 * @brief Generates photons for one "step" and simulates propagation through the ice.
 * @param group the group of threads used to process one step (eg one warp)
 * @param step the step to be processed
 * @param sharedPhotonInitials shared memory to store photon initial conditions,
 *                              !! size needs to be same as number of threads in the group !!
 * @param numPhotonsInShared keeps track of the number of photons currently stored in shared memory
 *                           !! needs to live in shared memory itself !!   
 */
__device__ __forceinline__  void propGroup(cg::thread_block_tile<32> group, const I3CLSimStepCuda &step,
                          I3CLPhoton *sharedPhotonInitials, int& numPhotonsInShared,
                          uint32_t *hitIndex, const uint32_t maxHitIndex, I3CLSimPhotonCuda *__restrict__ outputPhotons,
                          const float* wlenLut, const float* zOffsetLut, const float* sharedScatteringLength, 
                          const float* sharedAbsorptionADust, const float* sharedAbsorptionDeltaTau, RngType& rng,
                          const unsigned short *numIndexStringSetPointer,
                          const float* wlenBiasLutPointer,
                          const unsigned short* geoCellIndex0Pointer, 
                          const unsigned short* geoCellIndex1Pointer, 
                          const unsigned char* geoStringInSetPointer, 
                          const unsigned short* geoLayerNumPointer, 
                          const float* geoLayerStartZPointer, 
                          const float* geoLayerHeightPointer,
                          const float* geoStringPosXPointer, 
                          const float* geoStringPosYPointer,  
                          const float* geoStringMinZPointer, 
                          const float* geoStringMaxZPointer)
{
    // calculate step direction
    const float3 stepDir = calculateStepDir(step);

    // variables for managing shared memory
    int photonsLeftInStep = step.numPhotons; // will be 0 or negative if no photons are left

    // local variables for propagating the photon
    int photonId=-1; // threads with a photon id of 0 or bigger contain a valid photon
    I3CLPhoton photon; // this threads current photon

    // generate photon for every thread in the Warp from the step
    if(group.thread_rank() < photonsLeftInStep)
    {
        photon = createPhoton(step, stepDir, wlenLut, rng);
        photonId = 0; // set a valid id
    }
    photonsLeftInStep -= group.size(); // noet: if "photonsLeftInStep" goes negative, it does not matter 

    // make sure shared memory is not in use anymore from previous call
    group.sync();

    // generate photons and store in shared memory
    if(group.thread_rank() < photonsLeftInStep)    
        sharedPhotonInitials[group.thread_rank()] = createPhoton(step, stepDir, wlenLut, rng);
    if(group.thread_rank() == 0) 
    {
        float d = min(group.size(),photonsLeftInStep);
        numPhotonsInShared = d;
        photonsLeftInStep -= d;
    }
    photonsLeftInStep = group.shfl(photonsLeftInStep,0);
    group.sync();
    
    // loop as long as this thread has a valid photon, this is true for all threads while there is a photon left in the "step" or in shared memory
    while(photonId >= 0)
    {
        // propagate photon through the ice
        float distanceTraveled;
        bool absorbed = propPhoton(photon, distanceTraveled, rng, sharedScatteringLength, sharedAbsorptionADust, sharedAbsorptionDeltaTau, zOffsetLut);

        // check for collision with DOMs, if collision has happened, the hit will be stored in outputPhotons
        bool collided = checkForCollisionOld(photon, step, distanceTraveled, 
                                  hitIndex, maxHitIndex, outputPhotons, DOMOversizeFactor,
                                  numIndexStringSetPointer, wlenBiasLutPointer, 
                                  geoCellIndex0Pointer, geoCellIndex1Pointer, geoStringInSetPointer, geoLayerNumPointer, geoLayerStartZPointer, geoLayerHeightPointer,
                                  geoStringPosXPointer, geoStringPosYPointer, geoStringMinZPointer, geoStringMaxZPointer);

        if(collided || absorbed)
        {
            // photon is no longer valid
            photonId = -1;
        }
        else
        {
            // move the photon along its current direction for the distance it was propagated through the ice
            // then scatter to find a new direction vector
            updatePhotonTrack(photon, distanceTraveled);
            scatterPhoton(photon, rng);
        }

        if(numPhotonsInShared > 0 || photonsLeftInStep > 0)
        {
            // there are still photons waiting to be processed as long as this is true, all threads will be in the loop

            if(photonId < 0)
            {
                // try to grab a new photon from shared memory
                photonId = atomicAdd(&numPhotonsInShared,-1)-1;
                if(photonId >= 0)
                    photon = sharedPhotonInitials[photonId];
            }

            // if shared memory is empty, create new photons from the "step" (this branch is taken by all or none of the threads)
            group.sync(); // make sure all threads see the same value of numPhotonsInShared
            if( numPhotonsInShared <= 0 && photonsLeftInStep > 0)
            {
                if(group.thread_rank() < photonsLeftInStep)    
                    sharedPhotonInitials[group.thread_rank()] = createPhoton(step, stepDir, wlenLut, rng);
                    if(group.thread_rank() == 0) 
                    {
                        float d = min(group.size(),photonsLeftInStep);
                        numPhotonsInShared = d;
                        photonsLeftInStep -= d;
                    }
                    photonsLeftInStep = group.shfl(photonsLeftInStep,0);
                    group.sync();

                // if the thread did not have a valid photon, try again to get one now
                if(photonId < 0)
                {
                    // try to grab a new photon from shared memory
                    photonId = atomicAdd(&numPhotonsInShared,-1)-1;
                    if(photonId >= 0)
                        photon = sharedPhotonInitials[photonId];
                }
                group.sync(); // make sure all threads see the same value of numPhotonsInShared for the next iteration
            }
        }
    }
}

/**
 * @brief Main cuda kernel for photon propagation simulation. Called from host with a number of "steps".
 *        Photons for every "step" will be generated, propagated through the ice and stored if they hit the detector.
 *        Does some setup work. Then splits the current work group into thread groups.
 *        Each thread group simulated all photons in one step ( see propGroup() ).
 */
__global__ void propKernelJobqueue(I3CLSimStepCuda* __restrict__ steps, int numSteps, 
                            uint32_t* hitIndex, uint32_t maxHitIndex, I3CLSimPhotonCuda* __restrict__ outputPhotons,
                            const float* wlenLut, const float* zOffsetLut, 
                            uint64_t* __restrict__ rng_x, uint32_t* __restrict__ rng_a, int numPrimes)
{
    #ifdef SHARED_WLEN
        __shared__ float sharedWlenLut[WLEN_LUT_SIZE];
        for (int i = threadIdx.x; i < WLEN_LUT_SIZE; i += blockDim.x) {
            sharedWlenLut[i] = wlenLut[i];
        }
        const float* wlenLutPointer = sharedWlenLut;
    #else
        const float* wlenLutPointer = wlenLut;
    #endif

    #ifdef SHARED_ICE_PROPERTIES
        __shared__ float sharedScatteringLength[171];
        __shared__ float sharedAbsorptionADust[171];
        __shared__ float sharedAbsorptionDeltaTau[171];
        for (int i = threadIdx.x; i < 171; i += blockDim.x) {
            sharedScatteringLength[i] = scatteringLength_b400_LUT[i];
            sharedAbsorptionADust[i] = absorptionLength_aDust400_LUT[i];
            sharedAbsorptionDeltaTau[i] = absorptionLength_deltaTau_LUT[i];
        }
        const float* scatteringLutPointer = sharedScatteringLength;
        const float* absorbtionLutPointer = sharedAbsorptionADust;
        const float* absorbtionDeltaTauLutPointer = sharedAbsorptionDeltaTau;
    #else
        const float* scatteringLutPointer = scatteringLength_b400_LUT;
        const float* absorbtionLutPointer = absorptionLength_aDust400_LUT;
        const float* absorbtionDeltaTauLutPointer = absorptionLength_deltaTau_LUT;
    #endif

    #ifdef SHARED_WLEN_BIAS
        __shared__ float getWavelengthBias_dataShared[43];
        for (int i = threadIdx.x; i < 43; i += blockDim.x) {
            getWavelengthBias_dataShared[i] = getWavelengthBias_data[i];
        }
        const float* wlenBiasLutPointer = getWavelengthBias_dataShared;
    #else
        const float* wlenBiasLutPointer = getWavelengthBias_data;
    #endif

    #ifdef SHARED_NUM_INDEX_STRING_SET
        __shared__ unsigned short geoLayerToOMNumIndexPerStringSetLocal[GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE];
        for (int i = threadIdx.x; i < GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE; i += blockDim.x) {
            geoLayerToOMNumIndexPerStringSetLocal[i] = geoLayerToOMNumIndexPerStringSet[i];
        }
        const unsigned short* numIndexStringSetPointer = geoLayerToOMNumIndexPerStringSetLocal;
    #else
        const unsigned short* numIndexStringSetPointer = geoLayerToOMNumIndexPerStringSet;
    #endif

    #ifdef SHARED_COLLISION_GRID_DATA
        __shared__ unsigned short geoCellIndex0shared[GEO_CELL_NUM_X_0 * GEO_CELL_NUM_Y_0];
        __shared__ unsigned short geoCellIndex1shared[GEO_CELL_NUM_X_1 * GEO_CELL_NUM_Y_1];
        for (int i = threadIdx.x; i < GEO_CELL_NUM_X_0 * GEO_CELL_NUM_Y_0; i += blockDim.x) {
            geoCellIndex0shared[i] = geoCellIndex_0[i];
        }
        for (int i = threadIdx.x; i < GEO_CELL_NUM_X_1 * GEO_CELL_NUM_Y_1; i += blockDim.x) {
            geoCellIndex1shared[i] = geoCellIndex_1[i];
        }
        const unsigned short* geoCellIndex0Pointer = geoCellIndex0shared;
        const unsigned short* geoCellIndex1Pointer = geoCellIndex1shared;
    #else
        const unsigned short* geoCellIndex0Pointer = geoCellIndex_0;
        const unsigned short* geoCellIndex1Pointer = geoCellIndex_1;
    #endif

    #ifdef SHARED_STRING_DATA
        __shared__ unsigned char geoStringInSetShared[NUM_STRINGS];
        __shared__ unsigned short geoLayerNumShared[GEO_LAYER_STRINGSET_NUM];
        __shared__ float geoLayerStartZShared[GEO_LAYER_STRINGSET_NUM];
        __shared__ float geoLayerHeightShared[GEO_LAYER_STRINGSET_NUM];
        for (int i = threadIdx.x; i < NUM_STRINGS; i += blockDim.x) {
            geoStringInSetShared[i] = geoStringInStringSet[i];
        }
        for (int i = threadIdx.x; i < GEO_LAYER_STRINGSET_NUM; i += blockDim.x) {
            geoLayerNumShared[i] = geoLayerNum[i];
            geoLayerStartZShared[i] = geoLayerStartZ[i];
            geoLayerHeightShared[i] = geoLayerHeight[i];
        }
        const unsigned char* geoStringInSetPointer = geoStringInSetShared;
        const unsigned short* geoLayerNumPointer = geoLayerNumShared;
        const float* geoLayerStartZPointer = geoLayerStartZShared;
        const float* geoLayerHeightPointer = geoLayerHeightShared;
    #else
        const unsigned char* geoStringInSetPointer = geoStringInStringSet;
        const unsigned short* geoLayerNumPointer = geoLayerNum;
        const float* geoLayerStartZPointer = geoLayerStartZ;
        const float* geoLayerHeightPointer = geoLayerHeight;
    #endif

    #ifdef SHARED_STRING_POSITIONS
        __shared__ float geoStringPosXShared[NUM_STRINGS];
        __shared__ float geoStringPosYShared[NUM_STRINGS];
        __shared__ float geoStringMinZShared[NUM_STRINGS];
        __shared__ float geoStringMaxZShared[NUM_STRINGS];
        for (int i = threadIdx.x; i < NUM_STRINGS; i += blockDim.x) {
            geoStringPosXShared[i] = geoStringPosX[i];
            geoStringPosYShared[i] = geoStringPosY[i];
            geoStringMinZShared[i] = geoStringMinZ[i];
            geoStringMaxZShared[i] = geoStringMaxZ[i];
        }
        const float* geoStringPosXPointer = geoStringPosXShared;
        const float* geoStringPosYPointer = geoStringPosYShared;
        const float* geoStringMinZPointer = geoStringMinZShared;
        const float* geoStringMaxZPointer = geoStringMaxZShared;
    #else
        const float* geoStringPosXPointer = geoStringPosX;
        const float* geoStringPosYPointer = geoStringPosY;
        const float* geoStringMinZPointer = geoStringMinZ;
        const float* geoStringMaxZPointer = geoStringMaxZ;
    #endif

    __syncthreads();

    // get thread id
    cg::thread_block block = cg::this_thread_block();
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // initialize rng
    RngType rng(rng_x[threadId],rng_a[threadId%numPrimes]);

    // setup shared memory to hold generated photon initial conditions
    __shared__ I3CLPhoton sharedPhotonInitials[NTHREADS_PER_BLOCK];
    __shared__ int numPhotonsInShared[NTHREADS_PER_BLOCK / 32];

    // split up into warps sized groups, each group simulates one step at a time
    cg::thread_block_tile<32> group = cg::tiled_partition<32>(block);
    I3CLPhoton* thisGroupSharedPhotonInitials = &sharedPhotonInitials[0] + (group.size() * group.meta_group_rank());
    const int globalWarpId = blockIdx.x * group.meta_group_size() + group.meta_group_rank();
    const int totalNumWarps = group.meta_group_size() * gridDim.x;

    for(int i = globalWarpId; i < numSteps; i += totalNumWarps)
    {
        const I3CLSimStepCuda step = steps[i];
        propGroup(group, step, thisGroupSharedPhotonInitials, numPhotonsInShared[group.meta_group_rank()], 
                    hitIndex, maxHitIndex, outputPhotons, 
                    wlenLutPointer, zOffsetLut, scatteringLutPointer,
                    absorbtionLutPointer, absorbtionDeltaTauLutPointer, rng,
                    numIndexStringSetPointer, wlenBiasLutPointer, 
                    geoCellIndex0Pointer, geoCellIndex1Pointer, geoStringInSetPointer, geoLayerNumPointer, geoLayerStartZPointer, geoLayerHeightPointer,
                    geoStringPosXPointer, geoStringPosYPointer, geoStringMinZPointer, geoStringMaxZPointer);
    }

    // store rng state
    rng_x[threadId] = rng.getState();
}
#endif // CUDA_VERSION >= 11000
