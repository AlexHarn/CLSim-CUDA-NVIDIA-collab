#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) 2020, Ramona Hohl, rhohl@nvidia.com

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include <propagationKernelSource.cuh>
#include <propagationKernelFunctions.cuh>


#define STATS_TIMERS


hipError_t gl_err;

#define CUDA_ERR_CHECK(e)              \
    if (hipError_t(e) != hipSuccess) \
        printf("!!! Cuda Error %s in line %d \n", hipGetErrorString(hipError_t(e)), __LINE__);
#define CUDA_CHECK_CALL                     \
    gl_err = hipGetLastError();            \
    if (hipError_t(gl_err) != hipSuccess) \
        printf("!!! Cuda Error %s in line %d \n", hipGetErrorString(hipError_t(gl_err)), __LINE__ - 1);


        
void statsToFile( int* counters, unsigned int n){
    const std::string filename = "/home/rhohl/IceCube/offline_production/build/stats.csv";
    std::cout<< " writing "<< n << " to file "<< filename<<std::endl;
    std::ofstream outputFile; outputFile.open (filename);
    for (unsigned int i = 0; i <  n; i++)
    {  
            outputFile <<counters[i] << std::endl;
    }
    outputFile.close();
}


// remark: ignored tabulate version, removed ifdef TABULATE
// also removed ifdef DOUBLEPRECISION.
// SAVE_PHOTON_HISTORY  and SAVE_ALL_PHOTONS are not define for now, i.e. commented out these snippets,
// s.t. it corresponds to the default contstructor of I3CLSimStepToPhotonConverterOpenCL

__global__ __launch_bounds__(NTHREADS_PER_BLOCK, 4) void propKernel(
    uint32_t* hitIndex,          // deviceBuffer_CurrentNumOutputPhotons
    const uint32_t maxHitIndex,  // maxNumOutputPhotons_
    const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet,
    const I3CLSimStepCuda* __restrict__ inputSteps,  // deviceBuffer_InputSteps
    int nsteps,
    I3CLSimPhotonCuda* __restrict__ outputPhotons,  // deviceBuffer_OutputPhotons

#ifdef SAVE_PHOTON_HISTORY
    float4* photonHistory,
#endif
    uint64_t* __restrict__ MWC_RNG_x, uint32_t* __restrict__ MWC_RNG_a
    #ifdef STATS_TIMERS 
    ,clock_t* timers , float* counters, int* perStepCounter
     #endif
    );

// maxNumbWOrkItems from  CL rndm arrays
void init_RDM_CUDA(int maxNumWorkitems, uint64_t* MWC_RNG_x, uint32_t* MWC_RNG_a, uint64_t** d_MWC_RNG_x,
                   uint32_t** d_MWC_RNG_a)
{
    CUDA_ERR_CHECK(hipMalloc(d_MWC_RNG_a, maxNumWorkitems * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMalloc(d_MWC_RNG_x, maxNumWorkitems * sizeof(uint64_t)));

    CUDA_ERR_CHECK(hipMemcpy(*d_MWC_RNG_a, MWC_RNG_a, maxNumWorkitems * sizeof(uint32_t), hipMemcpyHostToDevice));
    CUDA_ERR_CHECK(hipMemcpy(*d_MWC_RNG_x, MWC_RNG_x, maxNumWorkitems * sizeof(uint64_t), hipMemcpyHostToDevice));

    hipDeviceSynchronize();
    printf("RNG is set up on CUDA gpu %d. \n", maxNumWorkitems);
}

void launch_CudaPropogate(const I3CLSimStep* __restrict__ in_steps, int nsteps, const uint32_t maxHitIndex,
                          unsigned short* geoLayerToOMNumIndexPerStringSet, int ngeolayer,
                          I3CLSimPhotonSeries& outphotons, uint64_t* __restrict__ MWC_RNG_x,
                          uint32_t* __restrict__ MWC_RNG_a, int sizeRNG, float& totalCudaKernelTime
                     
                        )
{
    #ifdef STATS_TIMERS
      //  nsteps = 1;
    #endif 


    // set up congruental random number generator, reusing host arrays and randomService from
    // I3CLSimStepToPhotonConverterOpenCL setup.
    uint64_t* d_MWC_RNG_x;
    uint32_t* d_MWC_RNG_a;
    init_RDM_CUDA(sizeRNG, MWC_RNG_x, MWC_RNG_a, &d_MWC_RNG_x, &d_MWC_RNG_a);

    printf("nsteps total = %d but dividing into %d launches of max size %d \n", nsteps, 1, nsteps);
    unsigned short* d_geolayer;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_geolayer, ngeolayer * sizeof(unsigned short)));
    CUDA_ERR_CHECK(hipMemcpy(d_geolayer, geoLayerToOMNumIndexPerStringSet, ngeolayer * sizeof(unsigned short),
                              hipMemcpyHostToDevice));
    

    struct I3CLSimStepCuda* h_cudastep = (struct I3CLSimStepCuda*)malloc(nsteps * sizeof(struct I3CLSimStepCuda));

    for (int i = 0; i < nsteps; i++) {
        h_cudastep[i] = I3CLSimStep(in_steps[i]);
        h_cudastep[i].identifier = i;
    }

    #ifdef STATS_TIMERS

   // h_cudastep[0].numPhotons = 1;
    /*
    h_cudastep[0] = I3CLSimStep(in_steps[2649]);
    h_cudastep[0].identifier = 2649;
    for (int i = 0; i < NTHREADS_PER_BLOCK; i++) {
        h_cudastep[i] = I3CLSimStep(in_steps[2649]);
        h_cudastep[i].identifier = 2649;
        h_cudastep[i].numPhotons = 1;
    }
    */
 
    #endif 
    
    I3CLSimStepCuda* d_cudastep;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudastep, nsteps * sizeof(I3CLSimStepCuda)));
    CUDA_ERR_CHECK(hipMemcpy(d_cudastep, h_cudastep, nsteps * sizeof(I3CLSimStepCuda), hipMemcpyHostToDevice));

    uint32_t* d_hitIndex;
    uint32_t h_hitIndex[1];
    h_hitIndex[0] = 0;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_hitIndex, 1 * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMemcpy(d_hitIndex, h_hitIndex, 1 * sizeof(uint32_t), hipMemcpyHostToDevice));

    I3CLSimPhotonCuda* d_cudaphotons;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudaphotons, maxHitIndex * sizeof(I3CLSimPhotonCuda)));

    int numBlocks = (nsteps + NTHREADS_PER_BLOCK - 1) / NTHREADS_PER_BLOCK;
    printf("launching kernel propKernel<<< %d , %d >>>( .., nsteps=%d)  \n", numBlocks, NTHREADS_PER_BLOCK, nsteps);

    #ifdef STATS_TIMERS 
    int ntimers = 5;
    float* d_counters  ;
    clock_t* d_timers  ;
    int * d_perStepCounter;
    const  char *  measurementName[ntimers]; 
    measurementName[1]= "download Photonn and Rng";
    measurementName[2]= "prop Photon";
    measurementName[3]= "check collision";
    measurementName[4]= "while loop";
    measurementName[0] = "whole Kernel";
    
    clock_t timer[numBlocks*2*ntimers];
    float counters[numBlocks*ntimers];
    int perStepCounter[nsteps];

    hipMalloc((void **)&d_timers, sizeof(clock_t) * ntimers * numBlocks *2);
    hipMalloc((void **)&d_counters, sizeof(float) * numBlocks*ntimers);
    hipMalloc((void **)&d_perStepCounter, sizeof(int) * nsteps);

    #endif 


    std::chrono::time_point<std::chrono::system_clock> startKernel = std::chrono::system_clock::now();
    propKernel<<<numBlocks, NTHREADS_PER_BLOCK>>>(d_hitIndex, maxHitIndex, d_geolayer, d_cudastep, nsteps,
                                                  d_cudaphotons, d_MWC_RNG_x, d_MWC_RNG_a
                                                  #ifdef STATS_TIMERS 
                                                  ,d_timers, d_counters,d_perStepCounter
                                                   #endif
                                                );
    CUDA_CHECK_CALL
    CUDA_ERR_CHECK(hipDeviceSynchronize());
    std::chrono::time_point<std::chrono::system_clock> endKernel = std::chrono::system_clock::now();
    totalCudaKernelTime = std::chrono::duration_cast<std::chrono::milliseconds>(endKernel - startKernel).count();

    CUDA_ERR_CHECK(hipMemcpy(timer, d_timers, sizeof(clock_t) *ntimers* numBlocks * 2, hipMemcpyDeviceToHost));
    CUDA_ERR_CHECK(hipMemcpy(counters, d_counters, sizeof(float) *ntimers* numBlocks, hipMemcpyDeviceToHost));
    CUDA_ERR_CHECK(hipMemcpy(perStepCounter,d_perStepCounter , sizeof(int) *nsteps, hipMemcpyDeviceToHost));

    statsToFile(perStepCounter, nsteps);


       // Compute the difference between the last block end and the first block start.
       clock_t minStart[ntimers];
       clock_t maxEnd[ntimers];
       double avrgTime[ntimers];
       double avrgCounters[ntimers];
 
       for( int m = 0; m< 5; ++m)
       {
           minStart[m] = timer[m* numBlocks + 0];
            maxEnd[m]  = timer[m* numBlocks + numBlocks*ntimers];
            avrgTime[m] = (timer[m* numBlocks + numBlocks*ntimers]-timer[m* numBlocks + 0])/numBlocks; 
            avrgCounters[m] = counters[m* numBlocks + 0]/numBlocks;
            if( m == 2 or m ==3)   avrgTime[m] = double(  timer[m* numBlocks + 0] )/numBlocks; 
       }

       
   
       for (int i = 1; i < numBlocks; i++)
       {
        for( int m = 0; m< 5; ++m)
        {
       
          minStart[m] = timer[m* numBlocks + i] < minStart[m] ? timer[m* numBlocks + i] : minStart[m];
          maxEnd[m] = timer[numBlocks*ntimers+m* numBlocks + i] > maxEnd[m] ? timer[numBlocks*ntimers+m* numBlocks + i] : maxEnd[m];

           avrgTime[m] += double(timer[numBlocks*ntimers+m* numBlocks + i] - timer[m* numBlocks + i] )/numBlocks; 
           avrgCounters[m] += counters[m* numBlocks + i]/numBlocks;
         
           if( m == 2 or m ==3)   avrgTime[m] += double(  timer[m* numBlocks + i] )/numBlocks; 
        }

       }

       printf("clock64 Cycles for one thread, i.e. one step   :  \n" );
       for( int m = 0; m< 5; ++m){
           printf("counted   %f repetition of ' %s '  = %f  and max span %f \n", avrgCounters[m], measurementName[m], avrgTime[m], double(maxEnd[m]-minStart[m]));
       }
           //printf("counted   %f repetition of ' %s '  = %f \n", avrgCounters[m], measurementName[m], avrgTime[m] );
      
       

    CUDA_ERR_CHECK(hipMemcpy(h_hitIndex, d_hitIndex, 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));
    int numberPhotons = h_hitIndex[0];

    if (numberPhotons > maxHitIndex) {
        printf("Maximum number of photons exceeded, only receiving %" PRIu32 " of %" PRIu32 " photons", maxHitIndex,
               numberPhotons);
        numberPhotons = maxHitIndex;
    }

    // copy (max fo maxHitIndex) photons to host.
    struct I3CLSimPhotonCuda* h_cudaphotons =
        (struct I3CLSimPhotonCuda*)malloc(numberPhotons * sizeof(struct I3CLSimPhotonCuda));
    CUDA_ERR_CHECK(hipMemcpy(h_cudaphotons, d_cudaphotons, numberPhotons * sizeof(I3CLSimPhotonCuda), hipMemcpyDeviceToHost));

    outphotons.resize(numberPhotons);
    for (int i = 0; i < numberPhotons; i++) {
        outphotons[i] = h_cudaphotons[i].getI3CLSimPhoton();
    }

 
    free(h_cudastep);
    free(h_cudaphotons);
    hipFree(d_cudaphotons);
    hipFree(d_cudastep);
    hipFree(d_geolayer);
    hipFree(d_MWC_RNG_a);
    hipFree(d_MWC_RNG_x);
    hipFree(d_counters);
    hipFree(d_timers);
    printf("photon hits = %i from %i steps \n", numberPhotons, nsteps);
}

/**
 * @brief Creates a single photon to be propagated
 * @param step the step to create the photon from
 * @param stepDir step direction to create the photon ( calculated in propGroup() )
 * @param _generateWavelength_0distY data needed for wavelength selection (pass pointer to global or shared data)
 * @param _generateWavelength_0distYCumulative data needed for wavelength selection (pass pointer to global or shared data) 
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL)
 */
__device__ __forceinline__ I3CLInitialPhoton createPhoton(const I3CLSimStepCuda &step, float4 stepDir, float* _generateWavelength_0distY, float* _generateWavelength_0distYCumulative, RNG_ARGS)
{
    // create a new photon
    I3CLInitialPhoton ph;
    createPhotonFromTrack(step, stepDir, RNG_ARGS_TO_CALL, ph.posAndTime, ph.dirAndWlen, _generateWavelength_0distY, _generateWavelength_0distYCumulative);
    ph.invGroupvel = 1.f / (getGroupVelocity(0, ph.dirAndWlen.w));

    // set an initial absorption length
    ph.absLength = -logf(RNG_CALL_UNIFORM_OC);
    return ph;
}

/**
 * @brief  propgates a single photon
 * @param ph the photon to propagate
 * @param distancePropagated the distance the photon was propagated during this iteration
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL)
 * @return the propagated distance
 */
__device__ __forceinline__ bool propPhoton(I3CLPhoton& ph, float& distancePropagated, RNG_ARGS)
{ 
    const float effective_z = ph.posAndTime.z - getTiltZShift(ph.posAndTime);
    const int currentPhotonLayer = min(max(findLayerForGivenZPos(effective_z), 0), MEDIUM_LAYERS - 1);
    const float photon_dz = ph.dirAndWlen.z;

    // add a correction factor to the number of absorption lengths
    // abs_lens_left before the photon is absorbed. This factor will be
    // taken out after this propagation step. Usually the factor is 1
    // and thus has no effect, but it is used in a direction-dependent
    // way for our model of ice anisotropy.
    const float abs_len_correction_factor = getDirectionalAbsLenCorrFactor(ph.dirAndWlen);
    ph.absLength *= abs_len_correction_factor;

    // the "next" medium boundary (either top or bottom, depending on
    // step direction)
    float mediumBoundary = (photon_dz < ZERO)
                                ? (mediumLayerBoundary(currentPhotonLayer))
                                : (mediumLayerBoundary(currentPhotonLayer) + (float)MEDIUM_LAYER_THICKNESS);

     // track this thing to the next scattering point
    float scaStepLeft = -logf(RNG_CALL_UNIFORM_OC);

    float currentScaLen = getScatteringLength(currentPhotonLayer, ph.dirAndWlen.w);
    float currentAbsLen = getAbsorptionLength(currentPhotonLayer, ph.dirAndWlen.w);

    float ais = (photon_dz * scaStepLeft - ((mediumBoundary - effective_z)) / currentScaLen) *
                (ONE / (float)MEDIUM_LAYER_THICKNESS);
    float aia = (photon_dz * ph.absLength - ((mediumBoundary - effective_z)) / currentAbsLen) *
                (ONE / (float)MEDIUM_LAYER_THICKNESS);

    
    // propagate through layers
    int j = currentPhotonLayer;
    if (photon_dz < 0) {
        for (; (j > 0) && (ais < ZERO) && (aia < ZERO);
                mediumBoundary -= (float)MEDIUM_LAYER_THICKNESS,
                currentScaLen = getScatteringLength(j, ph.dirAndWlen.w),
                currentAbsLen = getAbsorptionLength(j, ph.dirAndWlen.w), ais += 1.f / (currentScaLen),
                aia += 1.f / (currentAbsLen))
            --j;
    } else {
        for (; (j < MEDIUM_LAYERS - 1) && (ais > ZERO) && (aia > ZERO);
                mediumBoundary += (float)MEDIUM_LAYER_THICKNESS,
                currentScaLen = getScatteringLength(j, ph.dirAndWlen.w),
                currentAbsLen = getAbsorptionLength(j, ph.dirAndWlen.w), ais -= 1.f / (currentScaLen),
                aia -= 1.f / (currentAbsLen))
            ++j;
    }

    float distanceToAbsorption;
    if ((currentPhotonLayer == j) || ((my_fabs(photon_dz)) < EPSILON)) {
        distancePropagated = scaStepLeft * currentScaLen;
        distanceToAbsorption = ph.absLength * currentAbsLen;
    } else {
        const float recip_photon_dz = 1.f / (photon_dz);
        distancePropagated =
            (ais * ((float)MEDIUM_LAYER_THICKNESS) * currentScaLen + mediumBoundary - effective_z) *
            recip_photon_dz;
        distanceToAbsorption =
            (aia * ((float)MEDIUM_LAYER_THICKNESS) * currentAbsLen + mediumBoundary - effective_z) *
            recip_photon_dz;
    }

    // get overburden for distance i.e. check if photon is absorbed
    if (distanceToAbsorption < distancePropagated) {
        distancePropagated = distanceToAbsorption;
        ph.absLength = ZERO;
        return true;
    } else {
        ph.absLength = (distanceToAbsorption - distancePropagated) / currentAbsLen;
        
        // hoist the correction factor back out of the absorption length
        ph.absLength = ph.absLength / abs_len_correction_factor;
        return false;
    }

}

/**
 * @brief moves a photon along its track
 * @param ph the photon to move
 * @param distancePropagated the distance the photon was propagated this iteration
 */
__device__ __forceinline__  void updatePhotonTrack(I3CLPhoton& ph, float distancePropagated)
{
        ph.posAndTime.x += ph.dirAndWlen.x * distancePropagated;
        ph.posAndTime.y += ph.dirAndWlen.y * distancePropagated;
        ph.posAndTime.z += ph.dirAndWlen.z * distancePropagated;
        ph.posAndTime.w += ph.invGroupvel * distancePropagated;
        ph.totalPathLength += distancePropagated;
}

/**
 * @brief scatters a photon
 * @param ph the photon to scatter
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL) 
 */
__device__ __forceinline__  void scatterPhoton(I3CLPhoton& ph, RNG_ARGS)
{
     // optional direction transformation (for ice anisotropy)
    transformDirectionPreScatter(ph.dirAndWlen);

    // choose a scattering angle
    const float cosScatAngle = makeScatteringCosAngle(RNG_ARGS_TO_CALL);
    const float sinScatAngle = sqrt(ONE - sqr(cosScatAngle));

    // change the current direction by that angle
    scatterDirectionByAngle(cosScatAngle, sinScatAngle, ph.dirAndWlen, RNG_CALL_UNIFORM_CO);

    // optional direction transformation (for ice anisotropy)
    transformDirectionPostScatter(ph.dirAndWlen);

    ++ph.numScatters;
}

__global__ void propKernel(uint32_t* hitIndex,          // deviceBuffer_CurrentNumOutputPhotons
                           const uint32_t maxHitIndex,  // maxNumOutputPhotons_
                           const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet,
                           const I3CLSimStepCuda* __restrict__ inputSteps,  // deviceBuffer_InputSteps
                           int nsteps,
                           I3CLSimPhotonCuda* __restrict__ outputPhotons,  // deviceBuffer_OutputPhotons
                           uint64_t* __restrict__ MWC_RNG_x, uint32_t* __restrict__ MWC_RNG_a
                           #ifdef STATS_TIMERS 
                            ,clock_t* timers , float* counters, int* perStepCounter
                            #endif
                        )
{

    #ifdef STATS_TIMERS
        uint64_t start0, end0;
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;

        if(tid == 0){
           for (int m = 0; m<5; ++m) counters[m* gridDim.x + bid] = 0.0;
           for (int m = 0; m<5; ++m) timers[m* gridDim.x + bid] = 0.0;
         
       }
       int m = 0;

        if(tid == 0) start0 = clock64();

      

    #endif
  

#ifndef FUNCTION_getGroupVelocity_DOES_NOT_DEPEND_ON_LAYER
#error This kernel only works with a constant group velocity (constant w.r.t. layers)
#endif

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ unsigned short geoLayerToOMNumIndexPerStringSetLocal[GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE];
    __shared__ float _generateWavelength_0distYValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float _generateWavelength_0distYCumulativeValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float getWavelengthBias_dataShared[_generateWavelength_0NUM_DIST_ENTRIES];

    for (int ii = threadIdx.x; ii < GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE; ii += blockDim.x) {
        geoLayerToOMNumIndexPerStringSetLocal[ii] = geoLayerToOMNumIndexPerStringSet[ii];
    }

    for (int ii = threadIdx.x; ii < _generateWavelength_0NUM_DIST_ENTRIES; ii += blockDim.x) {
        _generateWavelength_0distYValuesShared[ii] = _generateWavelength_0distYValues[ii];
        _generateWavelength_0distYCumulativeValuesShared[ii] = _generateWavelength_0distYCumulativeValues[ii];
        getWavelengthBias_dataShared[ii] = getWavelengthBias_data[ii];
    }
     __syncthreads();
    if (i >= nsteps) return;

    
    #ifdef STATS_TIMERS
        uint64_t start, end;
        perStepCounter[i] = 0;
        if(tid == 0) start = clock64();
    #endif

    // download MWC RNG state
    uint64_t real_rnd_x = MWC_RNG_x[i];
    uint32_t real_rnd_a = MWC_RNG_a[i];
    uint64_t* rnd_x = &real_rnd_x;
    uint32_t* rnd_a = &real_rnd_a;

    const I3CLSimStepCuda step = inputSteps[i];

    #ifdef STATS_TIMERS
    if( tid==0 )
    {  
        end = clock64();
        m = 1;
        timers[m* gridDim.x + bid] = start;
        timers[m* gridDim.x + bid + 5*gridDim.x] = end;
        counters[m* gridDim.x + bid] += 1.0;
    }
   #endif


    float4 stepDir;
    {
        const float rho = sinf(step.dirAndLengthAndBeta.x);       // sin(theta)
        stepDir = float4{rho * cosf(step.dirAndLengthAndBeta.y),  // rho*cos(phi)
                         rho * sinf(step.dirAndLengthAndBeta.y),  // rho*sin(phi)
                         cosf(step.dirAndLengthAndBeta.x),        // cos(phi)
                         ZERO};
    }

 
     
    float nphot = step.numPhotons;
    uint32_t photonsLeftToPropagate = step.numPhotons;
    I3CLPhoton photon;
    photon.absLength = 0;
    I3CLInitialPhoton photonInitial;
    
 
    #ifdef STATS_TIMERS
        uint64_t start1, end1;
        if(tid == 0) start1 = clock64();
    #endif


   // #undef STATS_TIMERS
   
    while (photonsLeftToPropagate > 0) {
        
        if (photon.absLength < EPSILON) {
            photonInitial = createPhoton(step, stepDir,_generateWavelength_0distYValuesShared,_generateWavelength_0distYCumulativeValuesShared, RNG_ARGS_TO_CALL);
            photon = I3CLPhoton(photonInitial);
        }

        #ifdef STATS_TIMERS
             perStepCounter[i] += 1;
            if(tid == 0) start = clock64();
        #endif

        // this block is along the lines of the PPC kernel
        float distancePropagated;
        propPhoton(photon, distancePropagated, RNG_ARGS_TO_CALL);

        #ifdef STATS_TIMERS
        if( tid==0 )
        {  
            end = clock64();
            m =2;
            timers[m* gridDim.x + bid] += float(end- start);
            counters[m* gridDim.x + bid] += 1.0;
        }
      #endif


        #ifdef STATS_TIMERS
        if(tid == 0) start = clock64();
        #endif

        bool collided = checkForCollision(photon, photonInitial, step, distancePropagated, 
                                  hitIndex, maxHitIndex, outputPhotons, geoLayerToOMNumIndexPerStringSetLocal, getWavelengthBias_dataShared);

        #ifdef STATS_TIMERS
        if( tid==0 )
        {  
        
            end = clock64();
            m = 3;
            timers[m* gridDim.x + bid] += float(end- start);
            counters[m* gridDim.x + bid] += 1.0;
        }
       #endif

        if (collided) {
            // get rid of the photon if we detected it
            photon.absLength = ZERO;
     //       printf( " photon id = %u of global thread idx %u  and local thread idx %u and block idx %u \n" , photonsLeftToPropagate,i, threadIdx.x, blockIdx.x );
          
        }

        // absorb or scatter the photon
        if (photon.absLength < EPSILON) {
            // photon was absorbed.
            // a new one will be generated at the begin of the loop.
            --photonsLeftToPropagate;
        } else {  // photon was NOT absorbed. scatter it and re-start the loop

            updatePhotonTrack(photon, distancePropagated);
            scatterPhoton(photon, RNG_ARGS_TO_CALL);
        }
    }  // end while
   
    
    
    //#define STATS_TIMERS
    #ifdef STATS_TIMERS
    if( tid==0 )
    {  
       
        end1 = clock64();
        m = 4;
        timers[m* gridDim.x + bid] = start1;
        timers[m* gridDim.x + bid + 5*gridDim.x] = end1;
        counters[m* gridDim.x + bid] += 1.0;

        timers[2* gridDim.x + bid] =timers[2* gridDim.x + bid]/perStepCounter[i]*nphot ;
        timers[3* gridDim.x + bid] =timers[3* gridDim.x + bid]/perStepCounter[i]*nphot ;
    }
  #endif


    // upload MWC RNG state
    MWC_RNG_x[i] = real_rnd_x;
    MWC_RNG_a[i] = real_rnd_a;
    #define STATS_TIMERS
    #ifdef STATS_TIMERS
    if( tid==0 )
    {
          end0 = clock64();
          m = 0;
          timers[m* gridDim.x + bid] = start0;
          timers[m* gridDim.x + bid + 5*gridDim.x] = end0;
          counters[m* gridDim.x + bid] += 1.0;
        
   
    }
    perStepCounter[i] =  int(float(perStepCounter[i])/nphot);

    #endif
 
  
}
