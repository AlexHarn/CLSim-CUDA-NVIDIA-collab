#include "hip/hip_runtime.h"
/*The MIT License (MIT)

Copyright (c) 2020, Ramona Hohl, rhohl@nvidia.com

Permission is hereby granted, free of charge, to any person obtaining a copy of
this software and associated documentation files (the "Software"), to deal in
the Software without restriction, including without limitation the rights to
use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include <propagationKernelSource.cuh>
#include <propagationKernelFunctions.cuh>
#include <hiprand/hiprand_kernel.h>

hipError_t gl_err;

#define CUDA_ERR_CHECK(e)              \
    if (hipError_t(e) != hipSuccess) \
        printf("!!! Cuda Error %s in line %d \n", hipGetErrorString(hipError_t(e)), __LINE__);
#define CUDA_CHECK_CALL                     \
    gl_err = hipGetLastError();            \
    if (hipError_t(gl_err) != hipSuccess) \
        printf("!!! Cuda Error %s in line %d \n", hipGetErrorString(hipError_t(gl_err)), __LINE__ - 1);

// remark: ignored tabulate version, removed ifdef TABULATE
// also removed ifdef DOUBLEPRECISION.
// SAVE_PHOTON_HISTORY  and SAVE_ALL_PHOTONS are not define for now, i.e. commented out these snippets,
// s.t. it corresponds to the default contstructor of I3CLSimStepToPhotonConverterOpenCL

__global__ __launch_bounds__(NTHREADS_PER_BLOCK, 4) void propKernel(
    uint32_t* hitIndex,          // deviceBuffer_CurrentNumOutputPhotons
    const uint32_t maxHitIndex,  // maxNumOutputPhotons_
    const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet,
    const I3CLSimStepCuda* __restrict__ inputSteps,  // deviceBuffer_InputSteps
    int nsteps,
    I3CLSimPhotonCuda* __restrict__ outputPhotons,  // deviceBuffer_OutputPhotons

#ifdef SAVE_PHOTON_HISTORY
    float4* photonHistory,
#endif
    hiprandStatePhilox4_32_10_t* __restrict__ rngState);

// generates random state for hiprand
__global__ void generateRandomState(int seed, int numThreads, hiprandStatePhilox4_32_10_t* state)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id >= numThreads)
        return;
    hiprand_init(seed, id, 0, &state[id]);
}

void initRng(int numThreads, hiprandStatePhilox4_32_10_t** d_state, int seed = 161214)
{
    CUDA_ERR_CHECK(hipMalloc(d_state, numThreads * sizeof(hiprandStatePhilox4_32_10_t)));
    int numBlocks = (numThreads + NTHREADS_PER_BLOCK - 1) / NTHREADS_PER_BLOCK;
    generateRandomState<<<numBlocks, NTHREADS_PER_BLOCK>>>( seed, numThreads, *d_state);
    CUDA_ERR_CHECK(hipGetLastError());
}

void launch_CudaPropogate(const I3CLSimStep* __restrict__ in_steps, int nsteps, const uint32_t maxHitIndex,
                          unsigned short* geoLayerToOMNumIndexPerStringSet, int ngeolayer,
                          I3CLSimPhotonSeries& outphotons, uint64_t* __restrict__ MWC_RNG_x,
                          uint32_t* __restrict__ MWC_RNG_a, int sizeRNG, float& totalCudaKernelTime)
{
    // setup hiprand rng
    hiprandStatePhilox4_32_10_t* d_rngState;
    initRng(nsteps, &d_rngState);

    printf("nsteps total = %d but dividing into %d launches of max size %d \n", nsteps, 1, nsteps);
    unsigned short* d_geolayer;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_geolayer, ngeolayer * sizeof(unsigned short)));
    CUDA_ERR_CHECK(hipMemcpy(d_geolayer, geoLayerToOMNumIndexPerStringSet, ngeolayer * sizeof(unsigned short),
                              hipMemcpyHostToDevice));

    struct I3CLSimStepCuda* h_cudastep = (struct I3CLSimStepCuda*)malloc(nsteps * sizeof(struct I3CLSimStepCuda));

    for (int i = 0; i < nsteps; i++) {
        h_cudastep[i] = I3CLSimStep(in_steps[i]);
    }

    I3CLSimStepCuda* d_cudastep;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudastep, nsteps * sizeof(I3CLSimStepCuda)));
    CUDA_ERR_CHECK(hipMemcpy(d_cudastep, h_cudastep, nsteps * sizeof(I3CLSimStepCuda), hipMemcpyHostToDevice));

    uint32_t* d_hitIndex;
    uint32_t h_hitIndex[1];
    h_hitIndex[0] = 0;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_hitIndex, 1 * sizeof(uint32_t)));
    CUDA_ERR_CHECK(hipMemcpy(d_hitIndex, h_hitIndex, 1 * sizeof(uint32_t), hipMemcpyHostToDevice));

    I3CLSimPhotonCuda* d_cudaphotons;
    CUDA_ERR_CHECK(hipMalloc((void**)&d_cudaphotons, maxHitIndex * sizeof(I3CLSimPhotonCuda)));

    int numBlocks = (nsteps + NTHREADS_PER_BLOCK - 1) / NTHREADS_PER_BLOCK;
    printf("launching kernel propKernel<<< %d , %d >>>( .., nsteps=%d)  \n", numBlocks, NTHREADS_PER_BLOCK, nsteps);

    std::chrono::time_point<std::chrono::system_clock> startKernel = std::chrono::system_clock::now();
    propKernel<<<numBlocks, NTHREADS_PER_BLOCK>>>(d_hitIndex, maxHitIndex, d_geolayer, d_cudastep, nsteps,
                                                  d_cudaphotons, d_rngState);
    CUDA_ERR_CHECK(hipGetLastError());
    CUDA_ERR_CHECK(hipDeviceSynchronize());
    std::chrono::time_point<std::chrono::system_clock> endKernel = std::chrono::system_clock::now();
    totalCudaKernelTime = std::chrono::duration_cast<std::chrono::milliseconds>(endKernel - startKernel).count();

    CUDA_ERR_CHECK(hipMemcpy(h_hitIndex, d_hitIndex, 1 * sizeof(uint32_t), hipMemcpyDeviceToHost));
    int numberPhotons = h_hitIndex[0];

    if (numberPhotons > maxHitIndex) {
        printf("Maximum number of photons exceeded, only receiving %" PRIu32 " of %" PRIu32 " photons", maxHitIndex,
               numberPhotons);
        numberPhotons = maxHitIndex;
    }

    // copy (max fo maxHitIndex) photons to host.
    struct I3CLSimPhotonCuda* h_cudaphotons =
        (struct I3CLSimPhotonCuda*)malloc(numberPhotons * sizeof(struct I3CLSimPhotonCuda));
    CUDA_ERR_CHECK(
        hipMemcpy(h_cudaphotons, d_cudaphotons, numberPhotons * sizeof(I3CLSimPhotonCuda), hipMemcpyDeviceToHost));

    outphotons.resize(numberPhotons);
    for (int i = 0; i < numberPhotons; i++) {
        outphotons[i] = h_cudaphotons[i].getI3CLSimPhoton();
    }

    free(h_cudastep);
    free(h_cudaphotons);
    hipFree(d_cudaphotons);
    hipFree(d_cudastep);
    hipFree(d_geolayer);
    hipFree(d_rngState);
    printf("photon hits = %i from %i steps \n", numberPhotons, nsteps);
}

/**
 * @brief Creates a single photon to be propagated
 * @param step the step to create the photon from
 * @param stepDir step direction to create the photon ( calculated in propGroup() )
 * @param _generateWavelength_0distY data needed for wavelength selection (pass pointer to global or shared data)
 * @param _generateWavelength_0distYCumulative data needed for wavelength selection (pass pointer to global or shared data) 
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL)
 */
__device__ __forceinline__ I3CLInitialPhoton createPhoton(const I3CLSimStepCuda &step, float4 stepDir, float* _generateWavelength_0distY, float* _generateWavelength_0distYCumulative, RNG_ARGS)
{
    // create a new photon
    I3CLInitialPhoton ph;
    createPhotonFromTrack(step, stepDir, RNG_ARGS_TO_CALL, ph.posAndTime, ph.dirAndWlen, _generateWavelength_0distY, _generateWavelength_0distYCumulative);
    ph.invGroupvel = 1.f / (getGroupVelocity(0, ph.dirAndWlen.w));

    // set an initial absorption length
    ph.absLength = -logf(RNG_CALL_UNIFORM_OC);
    return ph;
}

/**
 * @brief  propgates a single photon
 * @param ph the photon to propagate
 * @param distancePropagated the distance the photon was propagated during this iteration
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL)
 * @return the propagated distance
 */
__device__ __forceinline__ bool propPhoton(I3CLPhoton& ph, float& distancePropagated, RNG_ARGS)
{ 
    const float effective_z = ph.posAndTime.z - getTiltZShift(ph.posAndTime);
    const int currentPhotonLayer = min(max(findLayerForGivenZPos(effective_z), 0), MEDIUM_LAYERS - 1);
    const float photon_dz = ph.dirAndWlen.z;

    // add a correction factor to the number of absorption lengths
    // abs_lens_left before the photon is absorbed. This factor will be
    // taken out after this propagation step. Usually the factor is 1
    // and thus has no effect, but it is used in a direction-dependent
    // way for our model of ice anisotropy.
    const float abs_len_correction_factor = getDirectionalAbsLenCorrFactor(ph.dirAndWlen);
    ph.absLength *= abs_len_correction_factor;

    // the "next" medium boundary (either top or bottom, depending on
    // step direction)
    float mediumBoundary = (photon_dz < ZERO)
                                ? (mediumLayerBoundary(currentPhotonLayer))
                                : (mediumLayerBoundary(currentPhotonLayer) + (float)MEDIUM_LAYER_THICKNESS);

     // track this thing to the next scattering point
    float scaStepLeft = -logf(RNG_CALL_UNIFORM_OC);

    float currentScaLen = getScatteringLength(currentPhotonLayer, ph.dirAndWlen.w);
    float currentAbsLen = getAbsorptionLength(currentPhotonLayer, ph.dirAndWlen.w);

    float ais = (photon_dz * scaStepLeft - ((mediumBoundary - effective_z)) / currentScaLen) *
                (ONE / (float)MEDIUM_LAYER_THICKNESS);
    float aia = (photon_dz * ph.absLength - ((mediumBoundary - effective_z)) / currentAbsLen) *
                (ONE / (float)MEDIUM_LAYER_THICKNESS);

    
    // propagate through layers
    int j = currentPhotonLayer;
    if (photon_dz < 0) {
        for (; (j > 0) && (ais < ZERO) && (aia < ZERO);
                mediumBoundary -= (float)MEDIUM_LAYER_THICKNESS,
                currentScaLen = getScatteringLength(j, ph.dirAndWlen.w),
                currentAbsLen = getAbsorptionLength(j, ph.dirAndWlen.w), ais += 1.f / (currentScaLen),
                aia += 1.f / (currentAbsLen))
            --j;
    } else {
        for (; (j < MEDIUM_LAYERS - 1) && (ais > ZERO) && (aia > ZERO);
                mediumBoundary += (float)MEDIUM_LAYER_THICKNESS,
                currentScaLen = getScatteringLength(j, ph.dirAndWlen.w),
                currentAbsLen = getAbsorptionLength(j, ph.dirAndWlen.w), ais -= 1.f / (currentScaLen),
                aia -= 1.f / (currentAbsLen))
            ++j;
    }

    float distanceToAbsorption;
    if ((currentPhotonLayer == j) || ((my_fabs(photon_dz)) < EPSILON)) {
        distancePropagated = scaStepLeft * currentScaLen;
        distanceToAbsorption = ph.absLength * currentAbsLen;
    } else {
        const float recip_photon_dz = 1.f / (photon_dz);
        distancePropagated =
            (ais * ((float)MEDIUM_LAYER_THICKNESS) * currentScaLen + mediumBoundary - effective_z) *
            recip_photon_dz;
        distanceToAbsorption =
            (aia * ((float)MEDIUM_LAYER_THICKNESS) * currentAbsLen + mediumBoundary - effective_z) *
            recip_photon_dz;
    }

    // get overburden for distance i.e. check if photon is absorbed
    if (distanceToAbsorption < distancePropagated) {
        distancePropagated = distanceToAbsorption;
        ph.absLength = ZERO;
        return true;
    } else {
        ph.absLength = (distanceToAbsorption - distancePropagated) / currentAbsLen;
        
        // hoist the correction factor back out of the absorption length
        ph.absLength = ph.absLength / abs_len_correction_factor;
        return false;
    }

}

/**
 * @brief moves a photon along its track
 * @param ph the photon to move
 * @param distancePropagated the distance the photon was propagated this iteration
 */
__device__ __forceinline__  void updatePhotonTrack(I3CLPhoton& ph, float distancePropagated)
{
        ph.posAndTime.x += ph.dirAndWlen.x * distancePropagated;
        ph.posAndTime.y += ph.dirAndWlen.y * distancePropagated;
        ph.posAndTime.z += ph.dirAndWlen.z * distancePropagated;
        ph.posAndTime.w += ph.invGroupvel * distancePropagated;
        ph.totalPathLength += distancePropagated;
}

/**
 * @brief scatters a photon
 * @param ph the photon to scatter
 * @param RNG_ARGS arguments for the random number generator (use RNG_ARGS_TO_CALL) 
 */
__device__ __forceinline__  void scatterPhoton(I3CLPhoton& ph, RNG_ARGS)
{
     // optional direction transformation (for ice anisotropy)
    transformDirectionPreScatter(ph.dirAndWlen);

    // choose a scattering angle
    const float cosScatAngle = makeScatteringCosAngle(RNG_ARGS_TO_CALL);
    const float sinScatAngle = sqrt(ONE - sqr(cosScatAngle));

    // change the current direction by that angle
    scatterDirectionByAngle(cosScatAngle, sinScatAngle, ph.dirAndWlen, RNG_CALL_UNIFORM_CO);

    // optional direction transformation (for ice anisotropy)
    transformDirectionPostScatter(ph.dirAndWlen);

    ++ph.numScatters;
}

__global__ void propKernel(uint32_t* hitIndex,          // deviceBuffer_CurrentNumOutputPhotons
                           const uint32_t maxHitIndex,  // maxNumOutputPhotons_
                           const unsigned short* __restrict__ geoLayerToOMNumIndexPerStringSet,
                           const I3CLSimStepCuda* __restrict__ inputSteps,  // deviceBuffer_InputSteps
                           int nsteps,
                           I3CLSimPhotonCuda* __restrict__ outputPhotons,  // deviceBuffer_OutputPhotons
                           hiprandStatePhilox4_32_10_t* __restrict__ rngState)
{
#ifndef FUNCTION_getGroupVelocity_DOES_NOT_DEPEND_ON_LAYER
#error This kernel only works with a constant group velocity (constant w.r.t. layers)
#endif

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ unsigned short geoLayerToOMNumIndexPerStringSetLocal[GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE];
    __shared__ float _generateWavelength_0distYValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float _generateWavelength_0distYCumulativeValuesShared[_generateWavelength_0NUM_DIST_ENTRIES];
    __shared__ float getWavelengthBias_dataShared[_generateWavelength_0NUM_DIST_ENTRIES];

    for (int ii = threadIdx.x; ii < GEO_geoLayerToOMNumIndexPerStringSet_BUFFER_SIZE; ii += blockDim.x) {
        geoLayerToOMNumIndexPerStringSetLocal[ii] = geoLayerToOMNumIndexPerStringSet[ii];
    }

    for (int ii = threadIdx.x; ii < _generateWavelength_0NUM_DIST_ENTRIES; ii += blockDim.x) {
        _generateWavelength_0distYValuesShared[ii] = _generateWavelength_0distYValues[ii];
        _generateWavelength_0distYCumulativeValuesShared[ii] = _generateWavelength_0distYCumulativeValues[ii];
        getWavelengthBias_dataShared[ii] = getWavelengthBias_data[ii];
    }
    __syncthreads();
    if (i >= nsteps) return;

    // download RNG state
    hiprandStatePhilox4_32_10_t real_thisRngState = rngState[i];
    hiprandStatePhilox4_32_10_t* thisRngState = &real_thisRngState;
    localRngData rngData;
    rngData.numRnums = 0;

    // printf("%f \n", RNG_CALL_UNIFORM_CO);

    const I3CLSimStepCuda step = inputSteps[i];
    float4 stepDir;
    {
        const float rho = sinf(step.dirAndLengthAndBeta.x);       // sin(theta)
        stepDir = float4{rho * cosf(step.dirAndLengthAndBeta.y),  // rho*cos(phi)
                         rho * sinf(step.dirAndLengthAndBeta.y),  // rho*sin(phi)
                         cosf(step.dirAndLengthAndBeta.x),        // cos(phi)
                         ZERO};
    }

    uint32_t photonsLeftToPropagate = step.numPhotons;
    I3CLPhoton photon;
    photon.absLength = 0;
    I3CLInitialPhoton photonInitial;

    while (photonsLeftToPropagate > 0) {
        if (photon.absLength < EPSILON) {
            photonInitial = createPhoton(step, stepDir,_generateWavelength_0distYValuesShared,_generateWavelength_0distYCumulativeValuesShared, RNG_ARGS_TO_CALL);
            photon = I3CLPhoton(photonInitial);
        }

        // this block is along the lines of the PPC kernel
        float distancePropagated;
        propPhoton(photon, distancePropagated, RNG_ARGS_TO_CALL);
        bool collided = checkForCollision(photon, photonInitial, step, distancePropagated, 
                                  hitIndex, maxHitIndex, outputPhotons, geoLayerToOMNumIndexPerStringSetLocal, getWavelengthBias_dataShared);

        if (collided) {
            // get rid of the photon if we detected it
            photon.absLength = ZERO;
        }

        // absorb or scatter the photon
        if (photon.absLength < EPSILON) {
            // photon was absorbed.
            // a new one will be generated at the begin of the loop.
            --photonsLeftToPropagate;
        } else {  // photon was NOT absorbed. scatter it and re-start the loop

            updatePhotonTrack(photon, distancePropagated);
            scatterPhoton(photon, RNG_ARGS_TO_CALL);
        }
    }  // end while

    // upload RNG state
    rngState[i] = real_thisRngState;
}
